#include "hip/hip_runtime.h"
#include "PyC_types.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "stdio.h"
#include <sys/time.h>
#define DEBUG 1
#define T 6
extern "C" {

// cleaner error handling; just wrap cuda library calls with gpuErrchk(foo());
#define gpuErr(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
} 

__global__
void reduction(double *H,double *R)
{
extern __shared__  double sdata[];
    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    sdata[tid] = H[i];

    __syncthreads();

    for (unsigned int s=blockDim.x/2; s>0; s>>=1)
    {
    if (tid<s)
    {
    sdata[tid]+=sdata[tid+s];
    }
    __syncthreads();
    }
    if (tid==0){R[blockIdx.x]=sdata[0];}
}

__global__
void outerloop(double *V1112,double *Rcca1,double *Ra2,double *H,int n_orb1,int n_orb2,double *V1222,double *Rcaa2, double *Rc1,double *h)
{
    int p1=threadIdx.x+blockIdx.x*blockDim.x;
    int q1=threadIdx.y+blockIdx.y*blockDim.y;
    int r1=threadIdx.z+blockIdx.z*blockDim.z;

    double Hlocal=0;

    if (p1 <n_orb1 && q1 <n_orb1 && r1<n_orb1)
    {	
        for (int s2=0;  s2<n_orb2;  s2++)
        {
    //upperloop
            Hlocal += V1112[((p1*n_orb1 + q1)*n_orb1 + r1)*n_orb2 + s2] * Rcca1[(q1*n_orb1 + p1)*n_orb1 + r1] * Ra2[s2];
    //middleloop
            Hlocal+=V1222[((p1*n_orb2 + q1)*n_orb2 + r1)*n_orb2 + s2] *     Rc1[p1] * Rcaa2[(q1*n_orb2 + s2)*n_orb2 + r1];
        
        }
    //bottomloop	
        H[(p1*n_orb1+q1)*n_orb1+r1]=2*Hlocal;
        H[(p1*n_orb1+q1)*n_orb1+r1]+=(r1==0)?(h[p1*n_orb2 + q1] * Rc1[p1] * Ra2[q1]):0;
    }
    //reduction still performed externally
}

void dimer_1min1pls_loop(PyInt n_elem, Double** H, PyInt* i, PyInt* j, PyInt* dim, PyFloat* sign, PyInt* n_orb1_in, PyInt* n_orb2_in, Double** Rc1,  Double** Rcca1, Double** Ra2, Double** Rcaa2, Double** h, Double** V1, Double** V2)
{
    struct timeval start,stop; 
    gettimeofday(&start,0);
    double *d_V1112,*d_Rcca1,*d_Ra2,*d_V1222,*d_Rcaa2,*d_Rc1,*d_H,*h_H,*d_h,*h_Hr,*d_Hr;

    // All of these assume that the values of n_orb1 and n_orb2 don't change
    const int n_orb1 =  n_orb1_in[0];
    const int n_orb2 =  n_orb2_in[0];
    const int blocks =  (n_orb1*n_orb1*n_orb1)/(T*T);
    const int nbpgrid=  n_orb1/T;
    const int N_V1112 = n_orb1*n_orb1*n_orb1*n_orb2;
    const int N_Rcca1 = n_orb1*n_orb1*n_orb1;
    const int N_Ra2 =   n_orb2;
    const int N_V1222 = n_orb1*n_orb2*n_orb2*n_orb2;
    const int N_Rcaa2 = n_orb2*n_orb2*n_orb2;
    const int N_Rc1 =   n_orb1;
    const int N_H =     n_orb1*n_orb1*n_orb1; // this assumes n_orb1 = n_orb2
    const int N_h =     n_orb1*n_orb2;
    const int N_Hr =    blocks;

    const dim3 dimblock(T,T,T);
    const dim3 dimgrid(nbpgrid,nbpgrid,nbpgrid);
    const dim3 dimblockR(T*T);
    const dim3 dimgridR(blocks);

    gpuErr(hipMalloc((void **) &d_V1112,   sizeof(double)*N_V1112));
    gpuErr(hipMalloc((void **) &d_Ra2,     sizeof(double)*N_Ra2));
    gpuErr(hipMalloc((void **) &d_V1222,   sizeof(double)*N_V1222));
    gpuErr(hipMalloc((void **) &d_Rcca1,   sizeof(double)*N_Rcca1));
    gpuErr(hipMalloc((void **) &d_Rcaa2,   sizeof(double)*N_Rcaa2));
    gpuErr(hipMalloc((void **) &d_Rc1,     sizeof(double)*N_Rc1));
    gpuErr(hipMalloc((void **) &d_H,       sizeof(double)*N_H));
    gpuErr(hipMalloc((void **) &d_h,       sizeof(double)*N_h));
    gpuErr(hipMalloc((void **) &d_Hr,      sizeof(double)*N_Hr));
    h_Hr=(double *)malloc(sizeof(double)*N_Hr);
    h_H=(double *)malloc(sizeof(double)*N_H);

    for(int n=0; n<n_elem; n++) {
        if(DEBUG) printf("n = %d\n", n);
        if(DEBUG) printf("n_orb1: %d, n_orb2: %d\n", n_orb1_in[n], n_orb2_in[n]);
        int index = i[n]*dim[n]+j[n];
        double tmp = 0.0;
        gpuErr(hipMemcpy(d_Rcca1,  Rcca1[n],   sizeof(double)*N_Rcca1, hipMemcpyHostToDevice));
        if(DEBUG) printf("Rcca1 copied\n");
        gpuErr(hipMemcpy(d_Rcaa2,  Rcaa2[n],   sizeof(double)*N_Rcaa2, hipMemcpyHostToDevice));
        if(DEBUG) printf("Rcaa2 copied\n");
        gpuErr(hipMemcpy(d_Rc1,    Rc1[n],     sizeof(double)*N_Rc1,   hipMemcpyHostToDevice));
        if(DEBUG) printf("Rc1 copied\n");
        gpuErr(hipMemcpy(d_Ra2,    Ra2[n],     sizeof(double)*N_Ra2,   hipMemcpyHostToDevice));
        if(DEBUG) printf("Ra2 copied\n");
        gpuErr(hipMemcpy(d_h,      h[n],       sizeof(double)*N_h,     hipMemcpyHostToDevice));
        if(DEBUG) printf("h copied\n");
        gpuErr(hipMemcpy(d_V1112,  V1[n],      sizeof(double)*N_V1112, hipMemcpyHostToDevice));
        if(DEBUG) printf("V1112 copied\n");
        gpuErr(hipMemcpy(d_V1222,  V2[n],      sizeof(double)*N_V1222, hipMemcpyHostToDevice));
        if(DEBUG) printf("V1222 copied\n");

        if(DEBUG) printf("Launching outerloop\n");
        outerloop<<<dimgrid,dimblock>>>(d_V1112,d_Rcca1,d_Ra2,d_H,n_orb1,n_orb2,d_V1222,d_Rcaa2,d_Rc1,d_h);
        gpuErr(hipPeekAtLastError());
        gpuErr(hipMemcpy(h_H, d_H, sizeof(double)*N_H, hipMemcpyDeviceToHost));
        gpuErr(hipDeviceSynchronize());
        if(DEBUG) printf("Launching reduction\n");
        reduction<<<dimgridR,dimblockR,sizeof(double)*T*T>>>(d_H,d_Hr);	
        gpuErr(hipPeekAtLastError());
        gpuErr(hipMemcpy(h_Hr, d_Hr, sizeof(double)*N_Hr, hipMemcpyDeviceToHost));
        gpuErr(hipDeviceSynchronize());
        for(int k=0; k<blocks; k++) {
            tmp += h_Hr[k];
        }
        H[n][index] = tmp;
    }
    gpuErr(hipFree(d_V1112));
    gpuErr(hipFree(d_Rcca1));
    gpuErr(hipFree(d_Ra2));
    gpuErr(hipFree(d_V1222));
    gpuErr(hipFree(d_Rcaa2));
    gpuErr(hipFree(d_Rc1));
    gpuErr(hipFree(d_H));
    gpuErr(hipFree(d_h));
    gpuErr(hipFree(d_Hr));
    free(h_Hr);
    free(h_H);

    gettimeofday(&stop,0);
    if(DEBUG) {
        double t = (double)(stop.tv_sec-start.tv_sec)*1000+(double)(stop.tv_usec-start.tv_usec)/1000;
        printf("dimer_1min1pls_loop inline cpu version finished in %f ms\n", t);
    }
}

}