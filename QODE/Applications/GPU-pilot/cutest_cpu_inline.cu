#include "PyC_types.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "stdio.h"
#include <sys/time.h>
#define DEBUG 1

extern "C" {


void dimer_1min1pls_loop(PyInt n_elem, Double** H, PyInt* i, PyInt* j, PyInt* dim, PyFloat* sign, PyInt* n_orb1, PyInt* n_orb2, Double** Rc1,  Double** Rcca1, Double** Ra2, Double** Rcaa2, Double** h, Double** V1, Double** V2)
{
    struct timeval start,stop; 
    gettimeofday(&start,0);
    for(int n=0; n<n_elem; n++) {
        int index = i[n]*dim[n]+j[n];
        double tmp = 0.0;

        // Upper loop
        for(int p1=0; p1<n_orb1[n]; p1++) {
            for(int r1=0; r1<n_orb1[n]; r1++) {
                for(int q1=0; q1<n_orb1[n]; q1++) {
                    for(int s2=0; s2<n_orb2[n]; s2++) {
                        tmp += V1[n][((p1*n_orb1[n] + q1)*n_orb1[n] + r1)*n_orb2[n] + s2] * Rcca1[n][(q1*n_orb1[n] + p1)*n_orb1[n] + r1] * Ra2[n][s2];
                    }
                }
            }
        }

        // Middle loop
        for(int p11=0; p11<n_orb1[n]; p11++) {
            for(int q2=0; q2<n_orb2[n]; q2++) {
                for(int r2=0; r2<n_orb2[n]; r2++) {
                    for(int s2=0; s2<n_orb2[n]; s2++) {
                        tmp += V2[n][((p11*n_orb2[n] + q2)*n_orb2[n] + r2)*n_orb2[n] + s2] * Rc1[n][p11] * Rcaa2[n][(q2*n_orb2[n] + s2)*n_orb2[n] + r2];
                    }
                }
            }
        }

        tmp *= 2;

        // Bottom Loop
        for(int p12=0; p12<n_orb1[n]; p12++) {
            for(int q22=0; q22<n_orb2[n]; q22++) {
                tmp += h[n][p12*n_orb2[n]+q22] * Rc1[n][p12] * Ra2[n][q22];
            }
        }
        H[n][index] = tmp*sign[n];
    }
    gettimeofday(&stop,0);
    if(DEBUG) {
        double t = (double)(stop.tv_sec-start.tv_sec)*1000+(double)(stop.tv_usec-start.tv_usec)/1000;
        printf("dimer_1min1pls_loop inline cpu version finished in %f ms\n", t);
    }
}

}