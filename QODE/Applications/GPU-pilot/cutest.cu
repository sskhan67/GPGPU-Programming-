#include "PyC_types.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "stdio.h"
#include <sys/time.h>
#include <iostream>

#define DEBUG 1
#define T 6

extern "C" {


void dimer_1min1pls_loop(PyInt n_elem, Double** H, PyInt* i, PyInt* j, PyInt* dim, PyFloat* sign, PyInt* n_orb1, PyInt* n_orb2, Double** Rc1,  Double** Rcca1, Double** Ra2, Double** Rcaa2, Double** h, Double** V1, Double** V2)
{
    struct timeval start,stop; 
    gettimeofday(&start,0);
    for(int n=0; n<n_elem; n++) {
        int index = i[n]*dim[n]+j[n];

        // Upper loop
        for(int p1=0; p1<n_orb1[n]; p1++) {
            for(int r1=0; r1<n_orb1[n]; r1++) {
                for(int q1=0; q1<n_orb1[n]; q1++) {
                    for(int s2=0; s2<n_orb2[n]; s2++) {
                        H[n][index] += sign[n] * V1[n][((p1*n_orb1[n] + q1)*n_orb1[n] + r1)*n_orb2[n] + s2] * Rcca1[n][(q1*n_orb1[n] + p1)*n_orb1[n] + r1] * Ra2[n][s2];
                    }
                }
            }
        }

        // Middle loop
        for(int p11=0; p11<n_orb1[n]; p11++) {
            for(int q2=0; q2<n_orb2[n]; q2++) {
                for(int r2=0; r2<n_orb2[n]; r2++) {
                    for(int s2=0; s2<n_orb2[n]; s2++) {
                        H[n][index] += sign[n] * V2[n][((p11*n_orb2[n] + q2)*n_orb2[n] + r2)*n_orb2[n] + s2] * Rc1[n][p1] * Rcaa2[n][(q2*n_orb2[n] + s2)*n_orb2[n] + r2];
                    }
                }
            }
        }

        H[n][index] *= 2;

        // Bottom Loop
        for(int p12=0; p12<n_orb1[n]; p12++) {
            for(int q22=0; q22<n_orb2[n]; q22++) {
                H[n][index] += h[n][p12*n_orb2[n]+q22] * Rc1[n][p12] * Ra2[q22];
            }
        }
    }
    gettimeofday(&stop,0);
    if(DEBUG) {
        double t = (double)(stop.tv_sec-start.tv_sec)*1000+(double)(stop.tv_usec-start.tv_usec)/1000;
        printf("dimer_1min1pls_loop finished in %f ms\n", t);
    }
}


}

 /*
    //std::vector<std::thread> hostThreads;
    // lambda expression to create streams
    auto createStreams = [streams]() {
        for(int i=0; i<numStreams; i++) {
            gpuErr(hipStreamCreate(&streams[i]));
        }
    };
   //hostThreads.push_back(std::move(std::thread(createStreams))); 
   for(std::thread& t : hostThreads) {
        if(t.joinable())
        t.join();  
    }
    */


    // not sure if hipMalloc is threadsafe if called from multiple host threads
    // will experiment with this -- concurrent allocation would save significant time
    // generalized lambda expression to perform a cudamalloc
    /*
    auto cudaPreMalloc = [](double *arr, unsigned long size) {
        gpuErr(hipMalloc((void **) &arr, size));
    };
    threads.push(std::thread(cudaPreMalloc, &d_V1112, sizeof(double)*N4*n_elem/numChunks));
    threads.push(std::thread(cudaPreMalloc, &d_Rcca1, sizeof(double)*N3*n_elem/numChunks));
    threads.push(std::thread(cudaPreMalloc, &d_Ra2,   sizeof(double)*n_orb2*n_elem/numChunks));
    threads.push(std::thread(cudaPreMalloc, &d_V1222, sizeof(double)*n_orb1*n_orb2*n_orb2*n_orb2*n_elem/numChunks));
    threads.push(std::thread(cudaPreMalloc, &d_Rcaa2, sizeof(double)*n_orb2*n_orb2*n_orb2*n_elem/numChunks));
    threads.push(std::thread(cudaPreMalloc, &d_Rc1,   sizeof(double)*n_orb1*n_elem/numChunks));
    threads.push(std::thread(cudaPreMalloc, &d_H,     sizeof(double)*n_orb1*n_orb1*n_orb1*n_elem/numChunks));
    threads.push(std::thread(cudaPreMalloc, &d_h,     sizeof(double)*n_orb1*n_orb2*n_elem/numChunks));
    threads.push(std::thread(cudaPreMalloc, &d_Hr,    sizeof(double)*blocks*n_elem/numChunks));
    */

