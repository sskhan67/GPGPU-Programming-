#include "hip/hip_runtime.h"
#include "PyC_types.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "stdio.h"
#include <sys/time.h>
#include <iostream>

#define PRINT_TIMES 1
#define PRINT_INSIDE_TIMES 0
#define DEBUG 1
#define T 6
#define numStreams 7
#define numChunks 8712 // copy data for 2 iterations at a time

extern "C" {

// cleaner error handling; just wrap cuda library calls with gpuErrchk(foo());
#define gpuErr(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
} 

__global__
void reduction(double *H,double *R)
{

extern __shared__  double sdata[];
    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    sdata[tid] = H[i];

    __syncthreads();

    for (unsigned int s=blockDim.x/2; s>0; s>>=1)
    {
    if (tid<s)
    {
    sdata[tid]+=sdata[tid+s];
    }
    __syncthreads();
    }
    if (tid==0){R[blockIdx.x]=sdata[0];}
}

__global__
void outerloop(double *V1112,double *Rcca1,double *Ra2,double *H,int n_orb1,int n_orb2,double *V1222,double *Rcaa2, double *Rc1,double *h)
{
    int p1=threadIdx.x+blockIdx.x*blockDim.x;
    int q1=threadIdx.y+blockIdx.y*blockDim.y;
    int r1=threadIdx.z+blockIdx.z*blockDim.z;

    double Hlocal=0;

    if (p1 <n_orb1 && q1 <n_orb1 && r1<n_orb1)
    {	
        for (int s2=0;  s2<n_orb2;  s2++)
        {
    //upperloop
            Hlocal += V1112[((p1*n_orb1 + q1)*n_orb1 + r1)*n_orb2 + s2] * Rcca1[(q1*n_orb1 + p1)*n_orb1 + r1] * Ra2[s2];
    //middleloop
            Hlocal+=V1222[((p1*n_orb2 + q1)*n_orb2 + r1)*n_orb2 + s2] *     Rc1[p1] * Rcaa2[(q1*n_orb2 + s2)*n_orb2 + r1];
        
        }
    //bottomloop	
        H[(p1*n_orb1+q1)*n_orb1+r1]=2*Hlocal;
        H[(p1*n_orb1+q1)*n_orb1+r1]+=(r1==0)?(h[p1*n_orb2 + q1] * Rc1[p1] * Ra2[q1]):0;
    }
    //reduction still performed externally
}

void dimer_1min1pls_loop(PyInt n_elem, Double** H, PyInt* i, PyInt* j, PyInt* dim, PyFloat* sign, PyInt* n_orb1_in, PyInt* n_orb2_in, Double** Rc1,  Double** Rcca1, Double** Ra2, Double** Rcaa2, Double** h, Double** V1, Double** V2)
{
    struct timeval start,stop; 
    gettimeofday(&start,0);

    hipStream_t streams[numStreams];
       for(int k=0; k<numStreams; k++) {
        gpuErr(hipStreamCreate(&streams[k]));
    }

    double *d_V1112,*d_Rcca1,*d_Ra2,*d_V1222,*d_Rcaa2,*d_Rc1,*d_H,*h_H,*d_h,*h_Hr,*d_Hr;

    // All of these assume that the values of n_orb1 and n_orb2 don't change
    const int n_orb1 = n_orb1_in[0];
    const int n_orb2 = n_orb1_in[0];
    const int blocks=(n_orb1*n_orb1*n_orb1)/(T*T);
    const int nbpgrid=n_orb1/T;
    const int N_V1112 = n_orb1*n_orb1*n_orb1*n_orb2;
    const int N_Rcca1 = n_orb1*n_orb1*n_orb1;
    const int N_Ra2 = n_orb2;
    const int N_V1222 = n_orb1*n_orb2*n_orb2*n_orb2;
    const int N_Rcaa2 = n_orb2*n_orb2*n_orb2;
    const int N_Rc1 = n_orb1;
    const int N_H = n_orb1*n_orb1*n_orb1; // this assumes n_orb1 = n_orb2
    const int N_h = n_orb1*n_orb2;
    const int N_Hr = blocks;


    if(n_elem%numChunks) {
        printf("Error in dimer_1min1pls: n_elem is not divisible by numChunks\n");
        exit(1);
    }
    const int iterationsPerChunk = n_elem / numChunks; // the above check guarantees that this integer division is safe
    if(DEBUG) printf("iterationsPerChunk: %d, numChunks: %d\n", iterationsPerChunk, numChunks);


    const dim3 dimblock(T,T,T);
    const dim3 dimgrid(nbpgrid,nbpgrid,nbpgrid);
    const dim3 dimblockR(T*T);
    const dim3 dimgridR(blocks);

    gpuErr(hipMalloc((void **) &d_V1112,   sizeof(double)*N_V1112*iterationsPerChunk));
    gpuErr(hipMalloc((void **) &d_Ra2,     sizeof(double)*N_Ra2*iterationsPerChunk));
    gpuErr(hipMalloc((void **) &d_V1222,   sizeof(double)*N_V1222*iterationsPerChunk));
    gpuErr(hipMalloc((void **) &d_Rcca1,   sizeof(double)*N_Rcca1*iterationsPerChunk));
    gpuErr(hipMalloc((void **) &d_Rcaa2,   sizeof(double)*N_Rcaa2*iterationsPerChunk));
    gpuErr(hipMalloc((void **) &d_Rc1,     sizeof(double)*N_Rc1*iterationsPerChunk));
    gpuErr(hipMalloc((void **) &d_H,       sizeof(double)*N_H*iterationsPerChunk));
    gpuErr(hipMalloc((void **) &d_h,       sizeof(double)*N_h*iterationsPerChunk));
    gpuErr(hipMalloc((void **) &d_Hr,      sizeof(double)*N_Hr*iterationsPerChunk));
    h_Hr=(double *)malloc(sizeof(double)*N_Hr);
    h_H=(double *)malloc(sizeof(double)*N_H);

    int chunk = 0;
    int count = iterationsPerChunk; // perform memcpy on the first iteration of the for loop
    for(int n=0; n<n_elem; n++) {
        if(count >= iterationsPerChunk) {
            count=0;
            if(n) { // not on the first iteration
                chunk++;
                if(chunk >= numChunks) {
                    break;
                }
            }
            /*
            // Synchronous 
            if(DEBUG) printf("Before Memcpy: Chunk: %d, Count: %d, n: %d\n", chunk,count,n);
            gpuErr(hipMemcpy(d_Rcca1,((&Rcca1[0][0])+(chunk*N_Rcca1*iterationsPerChunk)),sizeof(double)*N_Rcca1*iterationsPerChunk,hipMemcpyHostToDevice));
            if(DEBUG) printf("Rcca1 copied\n");
            gpuErr(hipMemcpy(d_Rcaa2,((&Rcaa2[0][0])+(chunk*N_Rcaa2*iterationsPerChunk)),sizeof(double)*N_Rcaa2*iterationsPerChunk,hipMemcpyHostToDevice));
            if(DEBUG) printf("Rcaa2 copied\n");
            gpuErr(hipMemcpy(d_Rc1,((&Rc1[0][0])+(chunk*N_Rc1*iterationsPerChunk)),sizeof(double)*N_Rc1*iterationsPerChunk,hipMemcpyHostToDevice));
            if(DEBUG) printf("Rc1 copied\n");
            gpuErr(hipMemcpy(d_Ra2,((&Ra2[0][0])+(chunk*N_Ra2*iterationsPerChunk)),sizeof(double)*N_Ra2*iterationsPerChunk,hipMemcpyHostToDevice));
            if(DEBUG) printf("Ra2 copied\n");
            gpuErr(hipMemcpy(d_h,((&h[0][0])+(chunk*N_h*iterationsPerChunk)),sizeof(double)*N_h*iterationsPerChunk,hipMemcpyHostToDevice));
            if(DEBUG) printf("h copied\n");
            gpuErr(hipMemcpy(d_V1112,((&V1[0][0])+(chunk*N_V1112*iterationsPerChunk)),sizeof(double)*N_V1112*iterationsPerChunk, hipMemcpyHostToDevice));
            if(DEBUG) printf("V1112 copied\n");
            gpuErr(hipMemcpy(d_V1222,((&V2[0][0])+(chunk*N_V1222*iterationsPerChunk)),sizeof(double)*N_V1222*iterationsPerChunk,hipMemcpyHostToDevice)); // v1222 memcpy segfaults on n>=86?
            if(DEBUG) printf("V1222 copied\n");
            */

            // Async
            gpuErr(hipMemcpyAsync(d_Rcca1, (&Rcca1[0][0]+(chunk*N_Rcca1*iterationsPerChunk)),  sizeof(double)*N_Rcca1*iterationsPerChunk,hipMemcpyHostToDevice, streams[0]));
            gpuErr(hipMemcpyAsync(d_Rcaa2, (&Rcaa2[0][0]+(chunk*N_Rcaa2*iterationsPerChunk)),  sizeof(double)*N_Rcaa2*iterationsPerChunk,hipMemcpyHostToDevice, streams[1]));
            gpuErr(hipMemcpyAsync(d_Rc1,   (&Rc1[0][0]+(chunk*N_Rc1*iterationsPerChunk)),      sizeof(double)*N_Rc1*iterationsPerChunk,hipMemcpyHostToDevice, streams[2]));
            gpuErr(hipMemcpyAsync(d_Ra2,   (&Ra2[0][0]+(chunk*N_Ra2*iterationsPerChunk)),      sizeof(double)*N_Ra2*iterationsPerChunk,hipMemcpyHostToDevice, streams[3]));
            gpuErr(hipMemcpyAsync(d_h,     (&h[0][0]+(chunk*N_h*iterationsPerChunk)),          sizeof(double)*N_h*iterationsPerChunk,hipMemcpyHostToDevice, streams[4]));
            gpuErr(hipMemcpyAsync(d_V1112, (&V1[0][0]+(chunk*N_V1112*iterationsPerChunk)),     sizeof(double)*N_V1112*iterationsPerChunk, hipMemcpyHostToDevice, streams[5]));
            gpuErr(hipMemcpyAsync(d_V1222, (&V2[0][0]+(chunk*N_V1222*iterationsPerChunk)),     sizeof(double)*N_V1222*iterationsPerChunk,hipMemcpyHostToDevice, streams[6])); 
            // v1222 memcpy segfaults on n>=86 for some reason?
            for(int s=0; s<numStreams;s++) {
                gpuErr(hipStreamSynchronize(streams[s]));
                if(DEBUG) printf("Stream %d synchronized\n", s);
            }
        }
        if(DEBUG) printf("Launching outerloop kernel\n");
        outerloop<<<dimgrid,dimblock>>>(d_V1112,d_Rcca1,d_Ra2,d_H,n_orb1,n_orb2,d_V1222,d_Rcaa2,d_Rc1,d_h);
        gpuErr(hipPeekAtLastError());
        gpuErr(hipMemcpyAsync(h_H,d_H,sizeof(double)*N_H,hipMemcpyDeviceToHost,streams[0]));
        gpuErr(hipDeviceSynchronize());
        if(DEBUG) printf("Launching reduction kernel\n");
        reduction<<<dimgridR,dimblockR,sizeof(double)*T*T>>>(d_H,d_Hr);	
        gpuErr(hipPeekAtLastError());
        gpuErr(hipMemcpyAsync(h_H,d_H,     sizeof(double)*N_H,hipMemcpyDeviceToHost,streams[0]));
        gpuErr(hipMemcpyAsync(h_Hr,d_Hr,   sizeof(double)*N_Hr,hipMemcpyDeviceToHost,streams[1]));
        gpuErr(hipStreamSynchronize(streams[0]));
        gpuErr(hipStreamSynchronize(streams[1]));
        double sum=0;
        for(int k=0;k<blocks;k++) {
            sum += h_Hr[k];
        }
       int index = i[n]*dim[n]*j[n];
        H[n][index] = sign[n] * sum;
        if(DEBUG) printf("Complete: Chunk: %d, Count: %d, n: %d\n", chunk,count,n);
        count ++;
    }

   // Cleanup
   for(int k=0; k<numStreams; k++) {
        gpuErr(hipStreamDestroy(streams[k]));
    }

    gpuErr(hipFree(d_V1112));
    gpuErr(hipFree(d_Rcca1));
    gpuErr(hipFree(d_Ra2));
    gpuErr(hipFree(d_V1222));
    gpuErr(hipFree(d_Rcaa2));
    gpuErr(hipFree(d_Rc1));
    gpuErr(hipFree(d_H));
    gpuErr(hipFree(d_h));
    gpuErr(hipFree(d_Hr));
    free(h_Hr);
    free(h_H);

    gettimeofday(&stop,0);
    if(DEBUG) {
        double t = (double)(stop.tv_sec-start.tv_sec)*1000+(double)(stop.tv_usec-start.tv_usec)/1000;
        printf("dimer_1min1pls_loop finished in %f ms\n", t);
    }
}


}

 /*
    //std::vector<std::thread> hostThreads;
    // lambda expression to create streams
    auto createStreams = [streams]() {
        for(int i=0; i<numStreams; i++) {
            gpuErr(hipStreamCreate(&streams[i]));
        }
    };
   //hostThreads.push_back(std::move(std::thread(createStreams))); 
   for(std::thread& t : hostThreads) {
        if(t.joinable())
        t.join();  
    }
    */


    // not sure if hipMalloc is threadsafe if called from multiple host threads
    // will experiment with this -- concurrent allocation would save significant time
    // generalized lambda expression to perform a cudamalloc
    /*
    auto cudaPreMalloc = [](double *arr, unsigned long size) {
        gpuErr(hipMalloc((void **) &arr, size));
    };
    threads.push(std::thread(cudaPreMalloc, &d_V1112, sizeof(double)*N4*n_elem/numChunks));
    threads.push(std::thread(cudaPreMalloc, &d_Rcca1, sizeof(double)*N3*n_elem/numChunks));
    threads.push(std::thread(cudaPreMalloc, &d_Ra2,   sizeof(double)*n_orb2*n_elem/numChunks));
    threads.push(std::thread(cudaPreMalloc, &d_V1222, sizeof(double)*n_orb1*n_orb2*n_orb2*n_orb2*n_elem/numChunks));
    threads.push(std::thread(cudaPreMalloc, &d_Rcaa2, sizeof(double)*n_orb2*n_orb2*n_orb2*n_elem/numChunks));
    threads.push(std::thread(cudaPreMalloc, &d_Rc1,   sizeof(double)*n_orb1*n_elem/numChunks));
    threads.push(std::thread(cudaPreMalloc, &d_H,     sizeof(double)*n_orb1*n_orb1*n_orb1*n_elem/numChunks));
    threads.push(std::thread(cudaPreMalloc, &d_h,     sizeof(double)*n_orb1*n_orb2*n_elem/numChunks));
    threads.push(std::thread(cudaPreMalloc, &d_Hr,    sizeof(double)*blocks*n_elem/numChunks));
    */

