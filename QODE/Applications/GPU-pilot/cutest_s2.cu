#include "hip/hip_runtime.h"
extern "C"
 {
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "stdlib.h"
#include "time.h"
#include "stdio.h"
#include "wrapper.h"
#include "PyC_types.h"
#include "stdlib.h"
#include <sys/time.h>
#define T 6


// Global variables
double *d_V1112,*d_Rcca1,*d_Ra2,*d_V1222,*d_Rcaa2,*d_Rc1;
double *h_H,*d_h;
double *d_H,*h_Hr,*d_Hr;





// working on it , sayed 
/*
__global__
void reduction(double *H,double *R)
{

extern __shared__ volatile double sdata[];
    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;
    sdata[tid] = H[i] + H[i+blockDim.x];
    
    __syncthreads();
    // do reduction in shared mem
    for (unsigned int s=blockDim.x/2; s>32; s>>=1) {
        if (tid < s) sdata[tid] += sdata[tid + s];
        __syncthreads();
    }

    if (tid < 32)
    {
        sdata[tid] += sdata[tid + 32];
        sdata[tid] += sdata[tid + 16];
        sdata[tid] += sdata[tid + 8];
        sdata[tid] += sdata[tid + 4];
        sdata[tid] += sdata[tid + 2];
        sdata[tid] += sdata[tid + 1];
    }
    // write result for this block to global mem
    if (tid == 0)
	{ 
	R[blockIdx.x] = sdata[0];
	//printf(" %lf", sdata[0]);
	}
}


*/

__global__
void reduction(double *H,double *R)
{

extern __shared__  double sdata[];
    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    sdata[tid] = H[i];

    __syncthreads();

	for (unsigned int s=blockDim.x/2; s>0; s>>=1)
	{
	if (tid<s)
	{
	sdata[tid]+=sdata[tid+s];
	}
	__syncthreads();
	}
	if (tid==0){R[blockIdx.x]=sdata[0];}
}








__global__
void outerloop(double *V1112,double *Rcca1,double *Ra2,double *H,int n_orb1,int n_orb2,double *V1222,double *Rcaa2, double *Rc1,double *h)
{
 int p1=threadIdx.x+blockIdx.x*blockDim.x;
 int q1=threadIdx.y+blockIdx.y*blockDim.y;
 int r1=threadIdx.z+blockIdx.z*blockDim.z;

 double Hlocal=0;

 if (p1 <n_orb1 && q1 <n_orb1 && r1<n_orb1)
 {	for (int s2=0;  s2<n_orb2;  s2++)
        {
//upperloop
	  Hlocal += V1112[((p1*n_orb1 + q1)*n_orb1 + r1)*n_orb2 + s2] * Rcca1[(q1*n_orb1 + p1)*n_orb1 + r1] * Ra2[s2];
//middleloop
	    Hlocal+=V1222[((p1*n_orb2 + q1)*n_orb2 + r1)*n_orb2 + s2] *     Rc1[p1] * Rcaa2[(q1*n_orb2 + s2)*n_orb2 + r1];
	  
        }
//bottomloop	
 	H[(p1*n_orb1+q1)*n_orb1+r1]=2*Hlocal;
        H[(p1*n_orb1+q1)*n_orb1+r1]+=(r1==0)?(h[p1*n_orb2 + q1] * Rc1[p1] * Ra2[q1]):0;
 }	
//reduction still performed externally

}

//exploring that n_orb1 and n_orb2 will be same


//TODO: Syed performs cudaMallocs just once (optimization 1, save and push file as cutest_s1.cu). Syed performs reduction inside kernel (optimization 2, which builds on 1,cutest_s2.cu).
//TODO: Thor performs cudastreams on this version and then later includes optimizations 1 and 2 from Syed. Thor also works on his own version for comparison.


//VKP: Skeleton code

double test_wrapper(int n_orb1, int n_orb2, double* Rc1, double* Rcca1,  double* Ra2,  double* Rcaa2,  double* h,  double* V1112,  double* V1222,int freevaribales)

{	


if (freevaribales==0) // will not execute hipFree and host free 
{

hipError_t cudaResult;
struct timeval start,stop,gpustart,gpustop;

int N4=n_orb1*n_orb1*n_orb1*n_orb2;
int N3=n_orb1*n_orb1*n_orb1;


int i=0;
gettimeofday(&gpustart,NULL);

	
dim3 dimblock(T,T,T);
int nbpgrid=n_orb1/T;
dim3 dimgrid(nbpgrid,nbpgrid,nbpgrid);
int blocks=(n_orb1*n_orb1*n_orb1)/(T*T);
dim3 dimblockR(T*T);
dim3 dimgridR(blocks);


	// counter-> assign dynamic memory only once at count=0
static int count=0;


if(count==0)
{

hipMalloc((void **)&d_V1112,sizeof(double)*N4);
hipMalloc((void **)&d_Rcca1,sizeof(double)*N3);
hipMalloc((void **)&d_Ra2,sizeof(double)*n_orb2);

hipMalloc((void **)&d_V1222,sizeof(double)*n_orb1*n_orb2*n_orb2*n_orb2);
hipMalloc((void **)&d_Rcaa2,sizeof(double)*n_orb2*n_orb2*n_orb2);
hipMalloc((void **)&d_Rc1,sizeof(double)*n_orb1);


h_Hr=(double *)malloc(sizeof(double)*blocks);
h_H=(double *)malloc(sizeof(double)*n_orb1*n_orb1*n_orb1);

hipMalloc((void **)&d_H,sizeof(double)*n_orb1*n_orb1*n_orb1);
hipMalloc((void **)&d_h,sizeof(double)*n_orb1*n_orb2);
hipMalloc((void **)&d_Hr,sizeof(double)*blocks);

}


gettimeofday(&start, NULL);

hipMemcpy(d_V1112,V1112,sizeof(double)*N4,hipMemcpyHostToDevice);
hipMemcpy(d_Rcca1,Rcca1,sizeof(double)*N3,hipMemcpyHostToDevice);
hipMemcpy(d_Ra2,Ra2,sizeof(double)*n_orb2,hipMemcpyHostToDevice);

hipMemcpy(d_V1222,V1222,sizeof(double)*n_orb1*n_orb2*n_orb2*n_orb2,hipMemcpyHostToDevice);
hipMemcpy(d_Rcaa2,Rcaa2,sizeof(double)*n_orb2*n_orb2*n_orb2,hipMemcpyHostToDevice);
hipMemcpy(d_Rc1,Rc1,sizeof(double)*n_orb1,hipMemcpyHostToDevice);
hipMemcpy(d_h,h,sizeof(double)*n_orb1*n_orb2,hipMemcpyHostToDevice);

cudaResult = hipGetLastError();
if (cudaResult != hipSuccess)
{
	printf("\n H2D failed...");
	printf("%s",hipGetErrorString(cudaResult));
}


 outerloop<<<dimgrid,dimblock>>>(d_V1112,d_Rcca1,d_Ra2,d_H,n_orb1,n_orb2,d_V1222,d_Rcaa2,d_Rc1,d_h);
 

cudaResult = hipGetLastError();
if (cudaResult != hipSuccess)
{
	printf("\n Outerloop failed...");
}



	// working on reduction-> sayed	

//printf("\n reduction threads:%d,blocks:%d",T*T,dimgridR.x);
	reduction<<<dimgridR,dimblockR,sizeof(double)*T*T>>>(d_H,d_Hr);	


 cudaResult = hipGetLastError();
if (cudaResult != hipSuccess)
{
	printf("\n reduction failed...");
}

	// external reduction  
hipMemcpyAsync(h_H,d_H,sizeof(double)*n_orb1*n_orb1*n_orb1,hipMemcpyDeviceToHost,streams[0]);
	// GPU reduction 
hipMemcpyAsync(h_Hr,d_Hr,sizeof(double)*blocks,hipMemcpyDeviceToHost,streams[1]);


 cudaStreamSynchonize(streams[0]);
 cudaStreamSynchonize(streams[1]);
if (cudaResult != hipSuccess)
{
	printf("\n D2H failed...");
}

	// gpu reduction 
double sum=0;
for(i=0;i<blocks;i++)
        sum+=h_Hr[i];

printf("Reduction  H: %lf \n", sum);


	// external reduction 

double H = 0;

  for(i=0;i<n_orb1*n_orb1*n_orb1;i++)
	H+=h_H[i];

printf("External reduction H : %lf \n",H);
printf("error: %lf \n", H-sum);

gettimeofday(&stop,NULL);
 double comptime=(double)(stop.tv_sec-start.tv_sec)*1000+(double)(stop.tv_usec-start.tv_usec)/1000;


gettimeofday(&gpustop,NULL);
double gputime=(double)(gpustop.tv_sec-gpustart.tv_sec)*1000+(double)(gpustop.tv_usec-gpustart.tv_usec)/1000;


 gettimeofday(&stop,NULL);

// double cpucomptime=(double)(stop.tv_sec-start.tv_sec)*1000+(double)(stop.tv_usec-start.tv_usec)/1000;

printf("\n GPU computation time: %lf ms GPU end to end: %lf ms",comptime,gputime);

count++;

return  H;

}


	// free dynamic memory 
else 
	{
	
	hipFree(d_V1112);
	hipFree(d_Rcca1);
	hipFree(d_Ra2);
	hipFree(d_V1222);
	hipFree(d_Rcaa2);
	hipFree(d_Rc1);
	hipFree(d_H);
	hipFree(d_h);
	hipFree(d_Hr);
	free(h_Hr);
	free(h_H);

	return 0;

	}





}




}
