#include "hip/hip_runtime.h"
/*   (C) Copyright 2018, 2020 Anthony D. Dutoi and Yuhong Liu
 *
 *   This file is part of Qode.
 *
 *   Qode is free software: you can redistribute it and/or modify
 *   it under the terms of the GNU General Public License as published by
 *   the Free Software Foundation, either version 3 of the License, or
 *   (at your option) any later version.
 *
 *   Qode is distributed in the hope that it will be useful,
 *   but WITHOUT ANY WARRANTY; without even the implied warranty of
 *   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *   GNU General Public License for more details.
 *
 *   You should have received a copy of the GNU General Public License
 *   along with Qode.  If not, see <http://www.gnu.org/licenses/>.
 */
#include "PyC_types.h"
#include<hip/hip_runtime.h>
//#include"wrapper.h"
#include "hip/hip_runtime.h"
#include "stdlib.h"
#include "time.h"
#include "stdio.h"
//#include "wrapper.h"
#include "PyC_types.h"
#include "stdlib.h"
#include <sys/time.h>
#include<omp.h>
#define T 6
#define TR 8
extern "C"
 {
/*
// Global variables
double *d_V1112,*d_Rcca1,*d_Ra2,*d_V1222,*d_Rcaa2,*d_Rc1;
double *h_H,*d_h;
double *d_H,*h_Hr,*d_Hr;
*/




// reduction works

__device__ void warpReduce(volatile double* sdata, int tid) {
sdata[tid] += sdata[tid + 32];
sdata[tid] += sdata[tid + 16];
sdata[tid] += sdata[tid + 8];
sdata[tid] += sdata[tid + 4];
sdata[tid] += sdata[tid + 2];
sdata[tid] += sdata[tid + 1];
}

__global__
void reduction(double *H,double *R, int numelements)
{
extern __shared__ volatile double sdata[];
    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;
if(i<numelements)
{
    sdata[tid] = H[i] + H[i+blockDim.x];
    
    __syncthreads();
    // do reduction in shared mem
    for (unsigned int s=blockDim.x/2; s>32; s>>=1) {
        //if (tid < s)
	{
	 sdata[tid] += sdata[tid + s];
        __syncthreads();
	}
	    }

	if (tid<32) warpReduce(sdata, tid);
	

  // write result for this block to global mem
    if (tid == 0)
	{ 
	R[blockIdx.x] = sdata[0];
	
	}
}
}










/*
__global__
void reduction(double *H,double *R, int numelements)
{

extern __shared__  double sdata[];
    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

 if (i<numelements)
        {
        sdata[tid] = H[i];

    __syncthreads();
        //if (i<numelements)
//      {
        for (unsigned int s=blockDim.x/2; s>0; s>>=1)
                {
                if (tid<s)
                {
                        sdata[tid]+=sdata[tid+s];
                }
                __syncthreads();
                }
        if (tid==0)
        {
        R[blockIdx.x]=sdata[0];
        }
        }
}


*/




__global__
void outerloop(double *V1112,double *Rcca1,double *Ra2,double *H,int n_orb1,int n_orb2,double *V1222,double *Rcaa2, double *Rc1,double *h)
{
 int p1=threadIdx.x+blockIdx.x*blockDim.x;
 int q1=threadIdx.y+blockIdx.y*blockDim.y;
 int r1=threadIdx.z+blockIdx.z*blockDim.z;

 double Hlocal=0;

 if (p1 <n_orb1 && q1 <n_orb1 && r1<n_orb1)
 {	for (int s2=0;  s2<n_orb2;  s2++)
        {
//upperloop
	  Hlocal += V1112[((p1*n_orb1 + q1)*n_orb1 + r1)*n_orb2 + s2] * Rcca1[(q1*n_orb1 + p1)*n_orb1 + r1] * Ra2[s2];
//middleloop
	    Hlocal+=V1222[((p1*n_orb2 + q1)*n_orb2 + r1)*n_orb2 + s2] *     Rc1[p1] * Rcaa2[(q1*n_orb2 + s2)*n_orb2 + r1];
	  
        }
//bottomloop	
 	H[(p1*n_orb1+q1)*n_orb1+r1]=2*Hlocal;
        H[(p1*n_orb1+q1)*n_orb1+r1]+=(r1==0)?(h[p1*n_orb2 + q1] * Rc1[p1] * Ra2[q1]):0;
 }	
//reduction still performed externally

}

//exploring that n_orb1 and n_orb2 will be same


//TODO: Syed performs cudaMallocs just once (optimization 1, save and push file as cutest_s1.cu). Syed performs reduction inside kernel (optimization 2, which builds on 1,cutest_s2.cu).
//TODO: Thor performs cudastreams on this version and then later includes optimizations 1 and 2 from Syed. Thor also works on his own version for comparison.


//VKP: Skeleton code

//double dimer_1min1pls(int n_orb1, int n_orb2, double* Rc1, double* Rcca1,  double* Ra2,  double* Rcaa2,  double* h,  double* V1112,  double* V1222,int freevaribales)
void dimer_1min1pls_loop(PyInt n_elem, Double** H, PyInt* i, PyInt* j, PyInt* dim, PyFloat* sign, PyInt* n_orb1_in, PyInt* n_orb2_in, Double** Rc1,  Double** Rcca1, Double** Ra2, Double** Rcaa2, Double** h, Double** V1, Double** V2)

{	
	//printf("hello");

        //hipMemcpy(d_h,      h[n],       sizeof(double)*N_h,     hipMemcpyHostToDevice);
        //if(DEBUG) printf("h copied\n");
	double *d_V1112,*d_Rcca1,*d_Ra2,*d_V1222,*d_Rcaa2,*d_Rc1,*d_H,*h_H,*d_h,*h_Hr,*d_Hr;

    // All of these assume that the values of n_orb1 and n_orb2 don't change
    const int n_orb1 =  n_orb1_in[0];
    const int n_orb2 =  n_orb2_in[0];
    const int blocks =  (n_orb1*n_orb1*n_orb1)/(TR*TR);
    const int nbpgrid=  n_orb1/T;
    const int N_V1112 = n_orb1*n_orb1*n_orb1*n_orb2;
    const int N_Rcca1 = n_orb1*n_orb1*n_orb1;
    const int N_Ra2 =   n_orb2;
    const int N_V1222 = n_orb1*n_orb2*n_orb2*n_orb2;
    const int N_Rcaa2 = n_orb2*n_orb2*n_orb2;
    const int N_Rc1 =   n_orb1;
    const int N_H =     n_orb1*n_orb1*n_orb1; // this assumes n_orb1 = n_orb2
    const int N_h =     n_orb1*n_orb2;
    const int N_Hr =    blocks;

    const dim3 dimblock(T,T,T);
    const dim3 dimgrid(nbpgrid,nbpgrid,nbpgrid);
    const dim3 dimblockR(T*T);
    const dim3 dimgridR(blocks);

    hipMalloc((void **) &d_V1112,   sizeof(double)*N_V1112);
    hipMalloc((void **) &d_Ra2,     sizeof(double)*N_Ra2);
    hipMalloc((void **) &d_V1222,   sizeof(double)*N_V1222);
    hipMalloc((void **) &d_Rcca1,   sizeof(double)*N_Rcca1);
    hipMalloc((void **) &d_Rcaa2,   sizeof(double)*N_Rcaa2);
    hipMalloc((void **) &d_Rc1,     sizeof(double)*N_Rc1);
    hipMalloc((void **) &d_H,       sizeof(double)*N_H);
    hipMalloc((void **) &d_h,       sizeof(double)*N_h);
    hipMalloc((void **) &d_Hr,      sizeof(double)*N_Hr);
    h_Hr=(double *)malloc(sizeof(double)*N_Hr);
    h_H=(double *)malloc(sizeof(double)*N_H);
    #pragma unroll 
  /*  
	hipMemcpy(d_Rcca1,  Rcca1,   sizeof(double)*N_Rcca1, hipMemcpyHostToDevice);
        hipMemcpy(d_Rcaa2,  Rcaa2,   sizeof(double)*N_Rcaa2, hipMemcpyHostToDevice);
        hipMemcpy(d_Rc1,    Rc1,     sizeof(double)*N_Rc1,   hipMemcpyHostToDevice);
        hipMemcpy(d_Ra2,    Ra2,     sizeof(double)*N_Ra2,   hipMemcpyHostToDevice);
        hipMemcpy(d_h,      h,       sizeof(double)*N_h,     hipMemcpyHostToDevice);
        hipMemcpy(d_V1112,  V1,      sizeof(double)*N_V1112, hipMemcpyHostToDevice);
        hipMemcpy(d_V1222,  V2,      sizeof(double)*N_V1222, hipMemcpyHostToDevice);
*/
    for(int n=0; n<n_elem; n++) {
        int index = i[n]*dim[n]+j[n];
        double tmp = 0.0;
        
	hipMemcpy(d_Rcca1,  Rcca1[n],   sizeof(double)*N_Rcca1, hipMemcpyHostToDevice);
        hipMemcpy(d_Rcaa2,  Rcaa2[n],   sizeof(double)*N_Rcaa2, hipMemcpyHostToDevice);
        hipMemcpy(d_Rc1,    Rc1[n],     sizeof(double)*N_Rc1,   hipMemcpyHostToDevice);
        hipMemcpy(d_Ra2,    Ra2[n],     sizeof(double)*N_Ra2,   hipMemcpyHostToDevice);
        hipMemcpy(d_h,      h[n],       sizeof(double)*N_h,     hipMemcpyHostToDevice);
        hipMemcpy(d_V1112,  V1[n],      sizeof(double)*N_V1112, hipMemcpyHostToDevice);
        hipMemcpy(d_V1222,  V2[n],      sizeof(double)*N_V1222, hipMemcpyHostToDevice);
       

	outerloop<<<dimgrid,dimblock>>>(d_V1112,d_Rcca1,d_Ra2,d_H,n_orb1,n_orb2,d_V1222,d_Rcaa2,d_Rc1,d_h);
        
	hipMemcpy(h_H, d_H, sizeof(double)*N_H, hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        //f(DEBUG) printf("Launching reduction\n");
        reduction<<<dimgridR,dimblockR,sizeof(double)*TR*TR>>>(d_H,d_Hr,(n_orb1*n_orb1*n_orb1));	
        //gpuErr(hipPeekAtLastError());
        hipMemcpy(h_Hr, d_Hr, sizeof(double)*N_Hr, hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        for(int k=0; k<blocks; k++) {
            tmp += h_Hr[k];
        }
        H[n][index] = tmp*sign[n];
    }
    hipFree(d_V1112);
    hipFree(d_Rcca1);
    hipFree(d_Ra2);
    hipFree(d_V1222);
    hipFree(d_Rcaa2);
    hipFree(d_Rc1);
    hipFree(d_H);
    hipFree(d_h);
    hipFree(d_Hr);
    free(h_Hr);
    free(h_H);

//printf("hello ");



}









Double monomer(PyInt n_orb, Double* Rca, Double* Rccaa, Double* h, Double* V)
	{
	Double H = 0;
	//printf("%d N orbibtal: ",n_orb);
	PyInt p=0;
	for (p=0;  p<n_orb;  p++)
		{
		PyInt q=0;
		for (q=0;  q<n_orb;  q++)
			{
			PyInt r=0;
			for ( r=0;  r<n_orb;  r++)
				{
				PyInt s=0;
				for (s=0;  s<n_orb;  s++)
					{
					H += V[((p*n_orb + q)*n_orb + r)*n_orb + s] * Rccaa[((p*n_orb + q)*n_orb + s)*n_orb + r];
					}
				}
			}
		}
	PyInt p1=0;
	for (p1=0;  p1<n_orb;  p1++)
		{
		PyInt q1=0;
		for (q1=0;  q1<n_orb;  q1++)
			{
			H += h[p1*n_orb + q1] * Rca[p1*n_orb + q1];
			}
		}
	return H;
	}



Double dimer_2min2pls(PyInt n_orb1, PyInt n_orb2, Double* Rcc1, Double* Raa2, Double* V)
	{
	Double H = 0;
	PyInt p1=0;
	for (p1=0;  p1<n_orb1;  p1++)
		{
		PyInt q1=0;
		for (q1=0;  q1<n_orb1;  q1++)
			{
			PyInt r2=0;
			for (r2=0;  r2<n_orb2;  r2++)
				{
				PyInt s2=0;
				for ( s2=0;  s2<n_orb2;  s2++)
					{
					H += V[((p1*n_orb1 + q1)*n_orb2 + r2)*n_orb2 + s2] * Rcc1[p1*n_orb1 + q1] * Raa2[s2*n_orb2 + r2];
					}
				}
			}
		}
	return H;
 
	}
/*
static int count=0;
Double dimer_1min1pls(PyInt n_orb1, PyInt n_orb2, Double* Rc1, Double* Rcca1, Double* Ra2, Double* Rcaa2, Double* h, Double* V1112, Double* V1222, int freevariables)
	{
	test_wrapper(n_orb1, n_orb2,  Rc1, Rcca1,  Ra2, Rcaa2,  h,  V1112,  V1222,freevariables);
	count++;
//if(count>10)
	//exit(0);

	}*/






/*

Double dimer_1min1pls(PyInt n_orb1, PyInt n_orb2, Double* Rc1, Double* Rcca1, Double* Ra2, Double* Rcaa2, Double* h, Double* V1112, Double* V1222)
	{
	Double H = 0;
	//printf("%d ord1: ",n_orb1);
	//printf(" n_orb2:%d\n ",n_orb2);
	//printf(" Rc1:%f \n",*Rc1);
	//printf(" Rcca1:%f\n ",*Rcca1);
	//printf(" Ra2: %f\n ",*Ra2);
	//printf(" h:%f",*h);
	//printf(" V1112:%f",* V1112);
	PyInt p1=0;
	for ( p1=0;  p1<n_orb1;  p1++)
		{
		PyInt q1=0;
		for ( q1=0;  q1<n_orb1;  q1++)
			{
			PyInt r1=0;
			for ( r1=0;  r1<n_orb1;  r1++)
				{
				PyInt s2=0;
				for ( s2=0;  s2<n_orb2;  s2++)
					{
					H += V1112[((p1*n_orb1 + q1)*n_orb1 + r1)*n_orb2 + s2] * Rcca1[(q1*n_orb1 + p1)*n_orb1 + r1] * Ra2[s2];
					}
				}
			}
		}
	PyInt p11=0;
	for (p11=0;  p11<n_orb1;  p11++)
		{
		PyInt q2=0;
		for (q2=0;  q2<n_orb2;  q2++)
			{
			PyInt r2=0;
			for (r2=0;  r2<n_orb2;  r2++)
				{
				PyInt s2=0;
				for ( s2=0;  s2<n_orb2;  s2++)
					{
					H += V1222[((p11*n_orb2 + q2)*n_orb2 + r2)*n_orb2 + s2] * Rc1[p1] * Rcaa2[(q2*n_orb2 + s2)*n_orb2 + r2];
					}
				}
			}
		}
	H *= 2;
	PyInt p12=0;
	for (p12=0;  p12<n_orb1;  p12++)
		{
		PyInt q22=0;
		for (q22=0;  q22<n_orb2;  q22++)
			{
			H += h[p12*n_orb2 + q22] * Rc1[p12] * Ra2[q22];
			}
		}
	//printf("%f H: ",H);
	return H;
	}

*/
Double dimer_00(PyInt n_orb1, PyInt n_orb2, Double* Rca1, Double* Rca2, Double* V)
	{
	Double H = 0;
	PyInt p1=0;
	for ( p1=0;  p1<n_orb1;  p1++)
		{
		PyInt q2=0;
		for ( q2=0;  q2<n_orb2;  q2++)
			{
			PyInt r1=0;
			for ( r1=0;  r1<n_orb1;  r1++)
				{
				PyInt s2=0;
				for ( s2=0;  s2<n_orb2;  s2++)
					{
					H += V[((p1*n_orb2 + q2)*n_orb1 + r1)*n_orb2 + s2] * Rca1[p1*n_orb1 + r1] * Rca2[q2*n_orb2 + s2];
					}
				}
			}
		}
	return 4*H;
	}



Double dimer_1pls1min(PyInt n_orb1, PyInt n_orb2, Double* Ra1, Double* Rcaa1, Double* Rc2, Double* Rcca2, Double* h, Double* V2221, Double* V2111)
	{
	Double H = 0;
	PyInt p2=0;
	for (p2=0;  p2<n_orb2;  p2++)
		{
		PyInt q2=0;
		for ( q2=0;  q2<n_orb2;  q2++)
			{
			PyInt r2=0;
			for ( r2=0;  r2<n_orb2;  r2++)
				{
				PyInt s1=0;
				for ( s1=0;  s1<n_orb1;  s1++)
					{
					H += V2221[((p2*n_orb2 + q2)*n_orb2 + r2)*n_orb1 + s1] * Rcca2[(q2*n_orb2 + p2)*n_orb2 + r2] * Ra1[s1];
					}
				}
			}
		}
	PyInt p22=0;
	for (p22=0;  p22<n_orb2;  p22++)
		{
		PyInt q1=0;
		for ( q1=0;  q1<n_orb1;  q1++)
			{
			PyInt r1=0;
			for (r1=0;  r1<n_orb1;  r1++)
				{
				PyInt s2=0;
				for (s2=0;  s2<n_orb1;  s2++)
					{
					H += V2111[((p22*n_orb1 + q1)*n_orb1 + r1)*n_orb1 + s2] * Rc2[p22] * Rcaa1[(q1*n_orb1 + s2)*n_orb1 + r1];
					}
				}
			}
		}
	H *= 2;
	PyInt p3=0;
	for (p3=0;  p3<n_orb2;  p3++)
		{
		PyInt q3=0;
		for (q3=0;  q3<n_orb1;  q3++)
			{
			H += h[p3*n_orb1 + q3] * Rc2[p3] * Ra1[q3];
			}
		}
	return H;
	}



Double dimer_2pls2min(PyInt n_orb1, PyInt n_orb2, Double* Raa1, Double* Rcc2, Double* V)
	{
	Double H = 0;
	PyInt p2=0;
	for (p2=0;  p2<n_orb2;  p2++)
		{
		PyInt q2=0;
		for (q2=0;  q2<n_orb2;  q2++)
			{
			PyInt r1=0;
			for (r1=0;  r1<n_orb1;  r1++)
				{
				PyInt s1=0;
				for (s1=0;  s1<n_orb1;  s1++)
					{
					H += V[((p2*n_orb2 + q2)*n_orb1 + r1)*n_orb1 + s1] * Rcc2[p2*n_orb2 + q2] * Raa1[s1*n_orb1 + r1];
					}
				}
			}
		}
	return H;
	}

/*

void dimer_1min1pls_loop(PyInt n_elem, Double** H, PyInt* i, PyInt* j, PyInt* dim, PyFloat* sign, PyInt* n_orb1, PyInt* n_orb2, Double** Rc1,  Double** Rcca1, Double** Ra2, Double** Rcaa2, Double** h, Double** V1, Double** V2)
{
    int n;
    for(n=0; n<n_elem; n++) {
        int index = i[n]*dim[n]+j[n];
        double tmp = 0.0;

        int p1,r1,q1,s2;
        for(p1=0; p1<n_orb1[n]; p1++) {
            for(r1=0; r1<n_orb1[n]; r1++) {
                for(q1=0; q1<n_orb1[n]; q1++) {
                    for( s2=0; s2<n_orb2[n]; s2++) {
                        tmp += V1[n][((p1*n_orb1[n] + q1)*n_orb1[n] + r1)*n_orb2[n] + s2] * Rcca1[n][(q1*n_orb1[n] + p1)*n_orb1[n] + r1] * Ra2[n][s2];
                    }
                }
            }
        }

     	int p11,q2,r2;
        for(p11=0; p11<n_orb1[n]; p11++) {
            for(q2=0; q2<n_orb2[n]; q2++) {
                for(r2=0; r2<n_orb2[n]; r2++) {
                    for(s2=0; s2<n_orb2[n]; s2++) {
                        tmp += V2[n][((p11*n_orb2[n] + q2)*n_orb2[n] + r2)*n_orb2[n] + s2] * Rc1[n][p11] * Rcaa2[n][(q2*n_orb2[n] + s2)*n_orb2[n] + r2];
                    }
                }
            }
        }

        tmp *= 2;

        int p12,q22;
        for(p12=0; p12<n_orb1[n]; p12++) {
            for(q22=0; q22<n_orb2[n]; q22++) {
                tmp += h[n][p12*n_orb2[n]+q22] * Rc1[n][p12] * Ra2[n][q22];
            }
        }
        H[n][index] = tmp*sign[n];
    }
    
    
}

*/

void monomer_loop(PyInt n_elem, Double** H, PyInt* i, PyInt* j, PyInt* dim, PyInt* n_orb, Double** Rca,  Double** Rccaa, Double** h, Double** V)
	{
	PyInt n=0;
	for ( n=0;  n<n_elem;  n++)
		{
		PyInt index = i[n]*dim[n] + j[n];
		H[n][index] = monomer(n_orb[n], Rca[n], Rccaa[n], h[n], V[n]);
		}
	return;
	}

void dimer_2min2pls_loop(PyInt n_elem, Double** H, PyInt* i, PyInt* j, PyInt* dim, PyInt* n_orb1, PyInt* n_orb2, Double** Rcc1, Double** Raa2, Double** V)
	{
	PyInt n=0;
	for ( n=0;  n<n_elem;  n++)
		{
		PyInt index = i[n]*dim[n] + j[n];
		H[n][index] = dimer_2min2pls(n_orb1[n], n_orb2[n], Rcc1[n], Raa2[n], V[n]);
		}
	return;
	}


/*

void dimer_1min1pls_loop(PyInt n_elem, Double** H, PyInt* i, PyInt* j, PyInt* dim, PyFloat* sign, PyInt* n_orb1, PyInt* n_orb2, Double** Rc1,  Double** Rcca1, Double** Ra2, Double** Rcaa2, Double** h, Double** V1, Double** V2)
	{

	int freevariables=0;
	PyInt n=0;
	#pragma unroll
	for ( n=0;  n<n_elem;  n++)
		{
		PyInt index = i[n]*dim[n] + j[n];
		H[n][index] = sign[n] * dimer_1min1pls(n_orb1[n], n_orb2[n], Rc1[n], Rcca1[n], Ra2[n], Rcaa2[n], h[n], V1[n], V2[n],freevariables);
		}
 	freevariables=1;
	dimer_1min1pls(n_orb1[n], n_orb2[n], Rc1[n], Rcca1[n], Ra2[n], Rcaa2[n], h[n], V1[n], V2[n],freevariables);
	return;
	}
*/



/*
void dimer_1min1pls_loop(PyInt n_elem, Double** H, PyInt* i, PyInt* j, PyInt* dim, PyFloat* sign, PyInt* n_orb1, PyInt* n_orb2, Double** Rc1,  Double** Rcca1, Double** Ra2, Double** Rcaa2, Double** h, Double** V1, Double** V2)
        {
	 test_wrapper( n_elem,H,i, j,dim,sign,n_orb1,n_orb2,Rc1,Rcca1,Ra2,Rcaa2, h,V1,V2);
	//return 
	}

*/
void dimer_00_loop(PyInt n_elem, Double** H, PyInt* i, PyInt* j, PyInt* dim, PyInt* n_orb1, PyInt* n_orb2, Double** Rca1, Double** Rca2, Double** V)
	{
	PyInt n=0;
	for ( n=0;  n<n_elem;  n++)
		{
		PyInt index = i[n]*dim[n] + j[n];
		H[n][index] = dimer_00(n_orb1[n], n_orb2[n], Rca1[n], Rca2[n], V[n]);
		}
	return;
	}

void dimer_1pls1min_loop(PyInt n_elem, Double** H, PyInt* i, PyInt* j, PyInt* dim, PyFloat* sign, PyInt* n_orb1, PyInt* n_orb2, Double** Ra1,  Double** Rcaa1, Double** Rc2, Double** Rcca2, Double** h, Double** V1, Double** V2)
	{
	PyInt n=0;
	for ( n=0;  n<n_elem;  n++)
		{
		PyInt index = i[n]*dim[n] + j[n];
		H[n][index] = sign[n] * dimer_1pls1min(n_orb1[n], n_orb2[n], Ra1[n], Rcaa1[n], Rc2[n], Rcca2[n], h[n], V1[n], V2[n]);
		}
	return;
	}

void dimer_2pls2min_loop(PyInt n_elem, Double** H, PyInt* i, PyInt* j, PyInt* dim, PyInt* n_orb1, PyInt* n_orb2, Double** Raa1, Double** Rcc2, Double** V)
	{
	PyInt n=0;
	for ( n=0;  n<n_elem;  n++)
		{
		PyInt index = i[n]*dim[n] + j[n];
		H[n][index] = dimer_2pls2min(n_orb1[n], n_orb2[n], Raa1[n], Rcc2[n], V[n]);
		}
	return;
	}


}
