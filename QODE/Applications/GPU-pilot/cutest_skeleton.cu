#include "hip/hip_runtime.h"
extern "C"
 {
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "stdlib.h"
#include "time.h"
#include "stdio.h"
#include "wrapper.h"
#include "PyC_types.h"
#include "stdlib.h"
#include <sys/time.h>
#define T 6
__global__
void reduction(double *H,double *R,int numele)
{
  __shared__ double partialSum[T*T];
  int t=threadIdx.x;
  int tid=threadIdx.x+blockIdx.x*blockDim.x;
 
  if(tid<numele) {
    partialSum[t]=H[tid];
 
    for(int stride=blockDim.x/2;stride>=1;stride/=2)
    {
    __syncthreads();
    if(t<stride)
          partialSum[t]+=partialSum[t+stride];
    } 
    if(t==0)
    R[blockIdx.x]=partialSum[t];
  } 
}
__global__
void outerloop(double *V1112,double *Rcca1,double *Ra2,double *H,int n_orb1,int n_orb2,double *V1222,double *Rcaa2, double *Rc1,double *h)
{
 int p1=threadIdx.x+blockIdx.x*blockDim.x;
 int q1=threadIdx.y+blockIdx.y*blockDim.y;
 int r1=threadIdx.z+blockIdx.z*blockDim.z;

 double Hlocal=0;

 if (p1 <n_orb1 && q1 <n_orb1 && r1<n_orb1)
 {	for (int s2=0;  s2<n_orb2;  s2++)
        {
//upperloop
      Hlocal += V1112[((p1*n_orb1 + q1)*n_orb1 + r1)*n_orb2 + s2] * Rcca1[(q1*n_orb1 + p1)*n_orb1 + r1] * Ra2[s2];
//middleloop
        Hlocal+=V1222[((p1*n_orb2 + q1)*n_orb2 + r1)*n_orb2 + s2] *     Rc1[p1] * Rcaa2[(q1*n_orb2 + s2)*n_orb2 + r1];
      
        }
//bottomloop	
     H[(p1*n_orb1+q1)*n_orb1+r1]=2*Hlocal;
        H[(p1*n_orb1+q1)*n_orb1+r1]+=(r1==0)?(h[p1*n_orb2 + q1] * Rc1[p1] * Ra2[q1]):0;
 }	
//reduction still performed externally

}

//exploring that n_orb1 and n_orb2 will be same


//TODO: Syed performs cudaMallocs just once (optimization 1, save and push file as cutest_s1.cu). Syed performs reduction inside kernel (optimization 2, which builds on 1,cutest_s2.cu).
//TODO: Thor performs cudastreams on this version and then later includes optimizations 1 and 2 from Syed. Thor also works on his own version for comparison.


//VKP: Skeleton code

double test_wrapper(int n_orb1, int n_orb2, double* Rc1, double* Rcca1,  double* Ra2,  double* Rcaa2,  double* h,  double* V1112,  double* V1222)

{	
//if(n_orb1!=18 && n_orb2!=18)
//	printf("\n Different values: %d %d",n_orb1,n_orb2);

hipError_t cudaResult;
struct timeval start,stop,gpustart,gpustop;
//printf("\n n_orb1:%d n_orb2:%d",n_orb1,n_orb2);

double *d_V1112,*d_Rcca1,*d_Ra2,*d_V1222,*d_Rcaa2,*d_Rc1;
double *d_H,*h_H,*d_h,*h_Hr,*d_Hr;

int N4=n_orb1*n_orb1*n_orb1*n_orb2;
int N3=n_orb1*n_orb1*n_orb1;
int i=0;
double H_gpu = 0.0;
gettimeofday(&gpustart,NULL);

dim3 dimblock(T,T,T);
int nbpgrid=n_orb1/T;
dim3 dimgrid(nbpgrid,nbpgrid,nbpgrid);
 int blocks=(n_orb1*n_orb1*n_orb1)/(T*T);
 dim3 dimblockR(T*T);
 dim3 dimgridR(blocks);


//Refactor such that these cudaMallocs are done just once because norbs never change

hipMalloc((void **)&d_V1112,sizeof(double)*N4);
hipMalloc((void **)&d_Rcca1,sizeof(double)*N3);
hipMalloc((void **)&d_Ra2,sizeof(double)*n_orb2);

hipMalloc((void **)&d_V1222,sizeof(double)*n_orb1*n_orb2*n_orb2*n_orb2);
hipMalloc((void **)&d_Rcaa2,sizeof(double)*n_orb2*n_orb2*n_orb2);
hipMalloc((void **)&d_Rc1,sizeof(double)*n_orb1);


 h_Hr=(double *)malloc(sizeof(double)*blocks);
 h_H=(double *)malloc(sizeof(double)*n_orb1*n_orb1*n_orb1);

hipMalloc((void **)&d_H,sizeof(double)*n_orb1*n_orb1*n_orb1);
hipMalloc((void **)&d_h,sizeof(double)*n_orb1*n_orb2);
hipMalloc((void **)&d_Hr,sizeof(double)*blocks);


gettimeofday(&start, NULL);

hipMemcpy(d_V1112,V1112,sizeof(double)*N4,hipMemcpyHostToDevice);
hipMemcpy(d_Rcca1,Rcca1,sizeof(double)*N3,hipMemcpyHostToDevice);
hipMemcpy(d_Ra2,Ra2,sizeof(double)*n_orb2,hipMemcpyHostToDevice);

hipMemcpy(d_V1222,V1222,sizeof(double)*n_orb1*n_orb2*n_orb2*n_orb2,hipMemcpyHostToDevice);
hipMemcpy(d_Rcaa2,Rcaa2,sizeof(double)*n_orb2*n_orb2*n_orb2,hipMemcpyHostToDevice);
hipMemcpy(d_Rc1,Rc1,sizeof(double)*n_orb1,hipMemcpyHostToDevice);
hipMemcpy(d_h,h,sizeof(double)*n_orb1*n_orb2,hipMemcpyHostToDevice);

cudaResult = hipGetLastError();
if (cudaResult != hipSuccess)
{
    printf("\n H2D failed...");
    printf("%s",hipGetErrorString(cudaResult));
}


 outerloop<<<dimgrid,dimblock>>>(d_V1112,d_Rcca1,d_Ra2,d_H,n_orb1,n_orb2,d_V1222,d_Rcaa2,d_Rc1,d_h);
 //middleloop<<<dimgrid,dimblock>>>(d_V1222,d_Rcaa2,d_Rc1,d_H,n_orb1,n_orb2);

cudaResult = hipGetLastError();
if (cudaResult != hipSuccess)
{
    printf("\n Outerloop failed...");
}

//printf("\n reduction threads:%d,blocks:%d",T*T,dimgridR.x);
//reduction<<<dimgridR,dimblockR>>>(d_H,d_Hr,n_orb1*n_orb1*n_orb1);
 cudaResult = hipGetLastError();
if (cudaResult != hipSuccess)
{
    printf("\n reduction failed...");
}
//hipMemcpy(h_Hr,d_Hr,sizeof(double)*blocks,hipMemcpyDeviceToHost);
 hipMemcpy(h_H,d_H,sizeof(double)*n_orb1*n_orb1*n_orb1,hipMemcpyDeviceToHost);
 hipDeviceSynchronize();

if (cudaResult != hipSuccess)
{
    printf("\n D2H failed...");
}
 

double H = 0;
//for(i=0;i<blocks;i++)
  for(i=0;i<n_orb1*n_orb1*n_orb1;i++)
    H_gpu +=h_H[i];

 
 gettimeofday(&stop,NULL);
 double comptime=(double)(stop.tv_sec-start.tv_sec)*1000+(double)(stop.tv_usec-start.tv_usec)/1000;

hipFree(d_V1112);
hipFree(d_Rcca1);
hipFree(d_Ra2);
hipFree(d_V1222);
hipFree(d_Rcaa2);
hipFree(d_Rc1);
hipFree(d_H);
hipFree(d_h);
hipFree(d_Hr);
free(h_Hr);
free(h_H);

gettimeofday(&gpustop,NULL);
double gputime=(double)(gpustop.tv_sec-gpustart.tv_sec)*1000+(double)(gpustop.tv_usec-gpustart.tv_usec)/1000;

printf("\n GPU computation time: %lf ms GPU end to end: %lf ms",comptime,gputime);
// printf("\n GPU printed: %lf in %lf ms",H,comptime);
gettimeofday(&start,NULL);
 H=0;	

    for (int p1=0;  p1<n_orb1;  p1++)
        {
        for (int q1=0;  q1<n_orb1;  q1++)
            {
            for (int r1=0;  r1<n_orb1;  r1++)
                {
                for (int s2=0;  s2<n_orb2;  s2++)
                    {
                    H += V1112[((p1*n_orb1 + q1)*n_orb1 + r1)*n_orb2 + s2] * Rcca1[(q1*n_orb1 + p1)*n_orb1 + r1] * Ra2[s2];
                    }
                }
            }
        }



for (int p1=0;  p1<n_orb1;  p1++)
        {
        for (int q2=0;  q2<n_orb2;  q2++)
            {
            for (int r2=0;  r2<n_orb2;  r2++)
                {
                for (int s2=0;  s2<n_orb2;  s2++)
                    {
                    H += V1222[((p1*n_orb2 + q2)*n_orb2 + r2)*n_orb2 + s2] * Rc1[p1] * Rcaa2[(q2*n_orb2 + s2)*n_orb2 + r2];
                    }
                }
            }
        }

    H *= 2;
    for (int p1=0;  p1<n_orb1;  p1++)
        {
        for (int q2=0;  q2<n_orb2;  q2++)
            {
            H += h[p1*n_orb2 + q2] * Rc1[p1] * Ra2[q2];
            }
        }

 gettimeofday(&stop,NULL);

  double cpucomptime=(double)(stop.tv_sec-start.tv_sec)*1000+(double)(stop.tv_usec-start.tv_usec)/1000;
// printf("\n CPU got: %lf in %lf ms",H,comptime);
printf("\n CPU: %lf GPU:%lf error: :%lf CPU time: %lf GPU time: %lf",H,H_gpu,(H_gpu-H),cpucomptime,comptime);
*/
return  H;


}




}
