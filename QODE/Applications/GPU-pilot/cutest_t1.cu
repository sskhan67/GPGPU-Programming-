#include "hip/hip_runtime.h"
//hipMalloc done just once. Streams only for memcpys. No reduction kerne


extern "C"
 {
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "stdlib.h"
#include "time.h"
#include "stdio.h"
#include "PyC_types.h"
#include <sys/time.h>
#define T 6
#define numStreams 7 // 7 for t1


// cleaner error handling; just wrap cuda library calls with gpuErrchk(foo());
#define gpuErr(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
} 


__global__
void reduction(double *H,double *R,int numele)
{
  __shared__ double partialSum[T*T];
  int t=threadIdx.x;
  int tid=threadIdx.x+blockIdx.x*blockDim.x;
 
  if(tid<numele) {
    partialSum[t]=H[tid];
 
    for(int stride=blockDim.x/2;stride>=1;stride/=2)
    {
	__syncthreads();
	if(t<stride)
  		partialSum[t]+=partialSum[t+stride];
    } 
    if(t==0)
	R[blockIdx.x]=partialSum[t];
  } 
}
__global__
void outerloop(double *V1112,double *Rcca1,double *Ra2,double *H,int n_orb1,int n_orb2,double *V1222,double *Rcaa2, double *Rc1,double *h)
{
 int p1=threadIdx.x+blockIdx.x*blockDim.x;
 int q1=threadIdx.y+blockIdx.y*blockDim.y;
 int r1=threadIdx.z+blockIdx.z*blockDim.z;

 double Hlocal=0;

 if (p1 <n_orb1 && q1 <n_orb1 && r1<n_orb1)
 {	for (int s2=0;  s2<n_orb2;  s2++)
        {
//upperloop
	  Hlocal += V1112[((p1*n_orb1 + q1)*n_orb1 + r1)*n_orb2 + s2] * Rcca1[(q1*n_orb1 + p1)*n_orb1 + r1] * Ra2[s2];
//middleloop
	    Hlocal+=V1222[((p1*n_orb2 + q1)*n_orb2 + r1)*n_orb2 + s2] *     Rc1[p1] * Rcaa2[(q1*n_orb2 + s2)*n_orb2 + r1];
	  
        }
//bottomloop	
 	H[(p1*n_orb1+q1)*n_orb1+r1]=2*Hlocal;
        H[(p1*n_orb1+q1)*n_orb1+r1]+=(r1==0)?(h[p1*n_orb2 + q1] * Rc1[p1] * Ra2[q1]):0;
 }	
//reduction still performed externally
}

//exploring that n_orb1 and n_orb2 will be same


//TODO: Syed performs cudaMallocs just once (optimization 1, save and push file as cutest_s1.cu). Syed performs reduction inside kernel (optimization 2, which builds on 1,cutest_s2.cu).
//TODO: Thor performs cudastreams on this version and then later includes optimizations 1 and 2 from Syed. Thor also works on his own version for comparison.

// create global streams once and re-use them in each call to test_wrapper_t1, then destroy them later (these are called in the loop function)


// t2:  break input into chunks
double test_wrapper_t2(int n_orb1, int n_orb2, double* Rc1, double* Rcca1,  double* Ra2,  double* Rcaa2,  double* h,  double* V1112,  double* V1222)
{	
double H = 0;
/*
double H_gpu = 0;
struct timeval start,stop,gpustart,gpustop;
//if(n_orb1!=18 && n_orb2!=18)
//	printf("\n Different values: %d %d",n_orb1,n_orb2);

hipError_t cudaResult;
//printf("\n n_orb1:%d n_orb2:%d",n_orb1,n_orb2);

double *d_V1112,*d_Rcca1,*d_Ra2,*d_V1222,*d_Rcaa2,*d_Rc1;
double *d_H,*h_H,*d_h,*h_Hr,*d_Hr;

int N4=n_orb1*n_orb1*n_orb1*n_orb2;
int N3=n_orb1*n_orb1*n_orb1;
int N2 = n_orb1 * n_orb2;
int i=0;
gettimeofday(&gpustart,NULL);
dim3 dimblock(T,T,T);

int nbpgrid=n_orb1/T;
dim3 dimgrid(numStreams/nbpgrid,numStreams/nbpgrid,numStreams/nbpgrid);
 int blocks=(n_orb1*n_orb1*n_orb1)/(T*T);
 dim3 dimblockR(T*T);
 dim3 dimgridR(blocks);


//Refactor such that these cudaMallocs are done just once because norbs never change

hipMalloc((void **)&d_V1112,sizeof(double)*N4);
hipMalloc((void **)&d_Rcca1,sizeof(double)*N3);
hipMalloc((void **)&d_Ra2,sizeof(double)*n_orb2);

hipMalloc((void **)&d_V1222,sizeof(double)*n_orb1*n_orb2*n_orb2*n_orb2);
hipMalloc((void **)&d_Rcaa2,sizeof(double)*n_orb2*n_orb2*n_orb2);
hipMalloc((void **)&d_Rc1,sizeof(double)*n_orb1);


 h_Hr=(double *)malloc(sizeof(double)*blocks);
 h_H=(double *)malloc(sizeof(double)*n_orb1*n_orb1*n_orb1);

hipMalloc((void **)&d_H,sizeof(double)*n_orb1*n_orb1*n_orb1);
hipMalloc((void **)&d_h,sizeof(double)*n_orb1*n_orb2);
hipMalloc((void **)&d_Hr,sizeof(double)*blocks);


gettimeofday(&start, NULL);

int s4 = N4 / numStreams;
int s3 = N3 / numStreams;
int s2 = N2 / numStreams;
hipMemcpyAsync(d_Ra2,Ra2,sizeof(double)*n_orb2,hipMemcpyHostToDevice, streams[0]);
hipMemcpyAsync(d_Rc1,Rc1,sizeof(double)*n_orb1,hipMemcpyHostToDevice, streams[1]);
for(int j=0; j<numStreams; j++) {
	int o4 = j*s4;
	int o3 = j*s3;
	int o2 = j*s2;
	hipMemcpyAsync(&d_V1112[o4], &V1112[o4], s4*sizeof(double), hipMemcpyHostToDevice, streams[j]);
	hipMemcpyAsync(&d_V1222[o4], &V1222[o4], s4*sizeof(double), hipMemcpyHostToDevice, streams[j]);
	hipMemcpyAsync(&d_Rcca1[o3], &Rcca1[o3], s3*sizeof(double), hipMemcpyHostToDevice, streams[j]);
	hipMemcpyAsync(&d_Rcaa2[o3] ,&Rcaa2[o3], s3*sizeof(double), hipMemcpyHostToDevice, streams[j]);
	hipMemcpyAsync(&d_h[o2],     &h[o2],     s2*sizeof(double), hipMemcpyHostToDevice, streams[j]);
}
for(int j=0; j<numStreams; j++) {
	int o4 = j*s4;
	int o3 = j*s3;
	int o2 = j*s2;
	outerloop<<<
}



cudaResult = hipGetLastError();
if (cudaResult != hipSuccess)
{
	printf("\n H2D failed...");
	printf("%s",hipGetErrorString(cudaResult));
}

 for(int j=0; j<numStreams; j++) {
    hipStreamSynchronize(streams[j]);
 }
 outerloop<<<dimgrid,dimblock>>>(d_V1112,d_Rcca1,d_Ra2,d_H,n_orb1,n_orb2,d_V1222,d_Rcaa2,d_Rc1,d_h);
 //middleloop<<<dimgrid,dimblock>>>(d_V1222,d_Rcaa2,d_Rc1,d_H,n_orb1,n_orb2);

cudaResult = hipGetLastError();
if (cudaResult != hipSuccess)
{
	printf("\n Outerloop failed...");
}

//printf("\n reduction threads:%d,blocks:%d",T*T,dimgridR.x);
//reduction<<<dimgridR,dimblockR>>>(d_H,d_Hr,n_orb1*n_orb1*n_orb1);
 cudaResult = hipGetLastError();
if (cudaResult != hipSuccess)
{
	printf("\n reduction failed...");
}
//hipMemcpy(h_Hr,d_Hr,sizeof(double)*blocks,hipMemcpyDeviceToHost);
 hipMemcpyAsync(h_H,d_H,sizeof(double)*n_orb1*n_orb1*n_orb1,hipMemcpyDeviceToHost, streams[0]);
 hipStreamSynchronize(streams[0]); // using hipDeviceSynchronize here is unnecessary

if (cudaResult != hipSuccess)
{
	printf("\n D2H failed...");
}
 

//for(i=0;i<blocks;i++)
  for(i=0;i<n_orb1*n_orb1*n_orb1;i++)
	H_gpu+=h_H[i];

 
 gettimeofday(&stop,NULL);

 double comptime=(double)(stop.tv_sec-start.tv_sec)*1000+(double)(stop.tv_usec-start.tv_usec)/1000;
hipFree(d_V1112);
hipFree(d_Rcca1);
hipFree(d_Ra2);
hipFree(d_V1222);
hipFree(d_Rcaa2);
hipFree(d_Rc1);
hipFree(d_H);
hipFree(d_h);
hipFree(d_Hr);
free(h_Hr);
free(h_H);
gettimeofday(&gpustop,NULL);
double gputime=(double)(gpustop.tv_sec-gpustart.tv_sec)*1000+(double)(gpustop.tv_usec-gpustart.tv_usec)/1000;
H = H_gpu;
printf("\n GPU computation time: %lf ms GPU end to end: %lf ms\n",comptime,gputime);
// printf("\n GPU printed: %lf in %lf ms",H,comptime);
gettimeofday(&start,NULL);
	for (int p1=0;  p1<n_orb1;  p1++)
		{
		for (int q1=0;  q1<n_orb1;  q1++)
			{
			for (int r1=0;  r1<n_orb1;  r1++)
				{
				for (int s2=0;  s2<n_orb2;  s2++)
					{
					H += V1112[((p1*n_orb1 + q1)*n_orb1 + r1)*n_orb2 + s2] * Rcca1[(q1*n_orb1 + p1)*n_orb1 + r1] * Ra2[s2];
					}
				}
			}
		}



for (int p1=0;  p1<n_orb1;  p1++)
		{
		for (int q2=0;  q2<n_orb2;  q2++)
			{
			for (int r2=0;  r2<n_orb2;  r2++)
				{
				for (int s2=0;  s2<n_orb2;  s2++)
					{
					H += V1222[((p1*n_orb2 + q2)*n_orb2 + r2)*n_orb2 + s2] * Rc1[p1] * Rcaa2[(q2*n_orb2 + s2)*n_orb2 + r2];
					}
				}
			}
		}

	H *= 2;
	for (int p1=0;  p1<n_orb1;  p1++)
		{
		for (int q2=0;  q2<n_orb2;  q2++)
			{
			H += h[p1*n_orb2 + q2] * Rc1[p1] * Ra2[q2];
			}
		}

 gettimeofday(&stop,NULL);

  double cpucomptime=(double)(stop.tv_sec-start.tv_sec)*1000+(double)(stop.tv_usec-start.tv_usec)/1000;
printf("\n CPU got: %lf in %lf ms",H,cpucomptime);
printf("\n CPU: %lf GPU:%lf error: :%lf CPU time: %lf GPU time: %lf",H,H_gpu,(H_gpu-H),cpucomptime,comptime);
*/
return  H;
}
int count =0;
void preMalloc(int n_orb1, int n_orb2)
{
	
}
// t1 and s1 : Asynchronous memcpys, 1 stream per input array, streams and mallocs done once
double test_wrapper(int n_orb1, int n_orb2, double* Rc1, double* Rcca1,  double* Ra2,  double* Rcaa2,  double* h,  double* V1112,  double* V1222, int freevariables)
{	
if(freevariables)
{
    for(int i=0; i<numStreams; i++) {
            hipStreamDestroy(streams[i]);
    }
    hipFree(d_V1112);
    hipFree(d_Rcca1);
    hipFree(d_Ra2);
    hipFree(d_V1222);
    hipFree(d_Rcaa2);
    hipFree(d_Rc1);
    hipFree(d_H);
    hipFree(d_h);
    hipFree(d_Hr);
    free(h_Hr);
    free(h_H);
    return 0.0;
}
if(!count) {
    preMalloc(n_orb1,n_orb2);
}
double H = 0.0;
struct timeval start,stop,gpustart,gpustop;
//if(n_orb1!=18 && n_orb2!=18)
//	printf("\n Different values: %d %d",n_orb1,n_orb2);

hipError_t cudaResult;
//printf("\n n_orb1:%d n_orb2:%d",n_orb1,n_orb2);

int i=0;
gettimeofday(&gpustart,NULL);
dim3 dimblock(T,T,T);
int N4=n_orb1*n_orb1*n_orb1*n_orb2;
int N3=n_orb1*n_orb1*n_orb1;
int nbpgrid=n_orb1/T;
dim3 dimgrid(nbpgrid,nbpgrid,nbpgrid);
dim3 dimblockR(T*T);
int blocks=(n_orb1*n_orb1*n_orb1)/(T*T);
dim3 dimgridR(blocks);


//Refactor such that these cudaMallocs are done just once because norbs never change
gettimeofday(&start, NULL);
hipMemcpyAsync(d_V1112,V1112,sizeof(double)*N4,hipMemcpyHostToDevice, streams[0]);
hipMemcpyAsync(d_Rcca1,Rcca1,sizeof(double)*N3,hipMemcpyHostToDevice, streams[1]);
hipMemcpyAsync(d_Ra2,Ra2,sizeof(double)*n_orb2,hipMemcpyHostToDevice, streams[2]);
hipMemcpyAsync(d_V1222,V1222,sizeof(double)*n_orb1*n_orb2*n_orb2*n_orb2,hipMemcpyHostToDevice, streams[3]);
hipMemcpyAsync(d_Rcaa2,Rcaa2,sizeof(double)*n_orb2*n_orb2*n_orb2,hipMemcpyHostToDevice, streams[4]);
hipMemcpyAsync(d_Rc1,Rc1,sizeof(double)*n_orb1,hipMemcpyHostToDevice, streams[5]);
hipMemcpyAsync(d_h,h,sizeof(double)*n_orb1*n_orb2,hipMemcpyHostToDevice, streams[6]);

cudaResult = hipGetLastError();
if (cudaResult != hipSuccess)
{
	printf("\n H2D failed...");
	printf("%s",hipGetErrorString(cudaResult));
}

 for(int j=0; j<numStreams; j++) {
    gpuErr(hipStreamSynchronize(streams[j]));
 }
 outerloop<<<dimgrid,dimblock>>>(d_V1112,d_Rcca1,d_Ra2,d_H,n_orb1,n_orb2,d_V1222,d_Rcaa2,d_Rc1,d_h);
 //middleloop<<<dimgrid,dimblock>>>(d_V1222,d_Rcaa2,d_Rc1,d_H,n_orb1,n_orb2);
 gpuErr(hipPeekAtLastError());

cudaResult = hipGetLastError();
if (cudaResult != hipSuccess)
{
	printf("\n Outerloop failed...");
}

//printf("\n reduction threads:%d,blocks:%d",T*T,dimgridR.x);
//reduction<<<dimgridR,dimblockR>>>(d_H,d_Hr,n_orb1*n_orb1*n_orb1);
 cudaResult = hipGetLastError();
if (cudaResult != hipSuccess)
{
	printf("\n reduction failed...");
}
//hipMemcpy(h_Hr,d_Hr,sizeof(double)*blocks,hipMemcpyDeviceToHost);
 hipMemcpyAsync(h_H,d_H,sizeof(double)*n_orb1*n_orb1*n_orb1,hipMemcpyDeviceToHost, streams[0]);
 hipDeviceSynchronize();

if (cudaResult != hipSuccess)
{
	printf("\n D2H failed...");
}
 

//for(i=0;i<blocks;i++)
  for(i=0;i<n_orb1*n_orb1*n_orb1;i++)
	H += h_H[i];

 
 gettimeofday(&stop,NULL);

 double comptime=(double)(stop.tv_sec-start.tv_sec)*1000+(double)(stop.tv_usec-start.tv_usec)/1000;

gettimeofday(&gpustop,NULL);
double gputime=(double)(gpustop.tv_sec-gpustart.tv_sec)*1000+(double)(gpustop.tv_usec-gpustart.tv_usec)/1000;
//H = H_gpu;
//printf("\n GPU computation time: %lf ms GPU end to end: %lf ms\n",comptime,gputime);
// printf("\n GPU printed: %lf in %lf ms",H,comptime);

count++;
return H;

}



}
