/*   (C) Copyright 2018, 2020 Anthony D. Dutoi and Yuhong Liu
 *
 *   This file is part of Qode.
 *
 *   Qode is free software: you can redistribute it and/or modify
 *   it under the terms of the GNU General Public License as published by
 *   the Free Software Foundation, either version 3 of the License, or
 *   (at your option) any later version.
 *
 *   Qode is distributed in the hope that it will be useful,
 *   but WITHOUT ANY WARRANTY; without even the implied warranty of
 *   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *   GNU General Public License for more details.
 *
 *   You should have received a copy of the GNU General Public License
 *   along with Qode.  If not, see <http://www.gnu.org/licenses/>.
 */
#include "PyC_types.h"
#include<hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include "stdlib.h"
#include "time.h"
#include "stdio.h"
#include "stdlib.h"
#include <sys/time.h>
#include<omp.h>

extern "C"
 {

Double monomer(PyInt n_orb, Double* Rca, Double* Rccaa, Double* h, Double* V)
	{
	Double H = 0;
	//printf("%d N orbibtal: ",n_orb);
	PyInt p=0;
	for (p=0;  p<n_orb;  p++)
		{
		PyInt q=0;
		for (q=0;  q<n_orb;  q++)
			{
			PyInt r=0;
			for ( r=0;  r<n_orb;  r++)
				{
				PyInt s=0;
				for (s=0;  s<n_orb;  s++)
					{
					H += V[((p*n_orb + q)*n_orb + r)*n_orb + s] * Rccaa[((p*n_orb + q)*n_orb + s)*n_orb + r];
					}
				}
			}
		}
	PyInt p1=0;
	for (p1=0;  p1<n_orb;  p1++)
		{
		PyInt q1=0;
		for (q1=0;  q1<n_orb;  q1++)
			{
			H += h[p1*n_orb + q1] * Rca[p1*n_orb + q1];
			}
		}
	return H;
	}



Double dimer_2min2pls(PyInt n_orb1, PyInt n_orb2, Double* Rcc1, Double* Raa2, Double* V)
	{
	Double H = 0;
	PyInt p1=0;
	for (p1=0;  p1<n_orb1;  p1++)
		{
		PyInt q1=0;
		for (q1=0;  q1<n_orb1;  q1++)
			{
			PyInt r2=0;
			for (r2=0;  r2<n_orb2;  r2++)
				{
				PyInt s2=0;
				for ( s2=0;  s2<n_orb2;  s2++)
					{
					H += V[((p1*n_orb1 + q1)*n_orb2 + r2)*n_orb2 + s2] * Rcc1[p1*n_orb1 + q1] * Raa2[s2*n_orb2 + r2];
					}
				}
			}
		}
	return H;
 
	}


Double dimer_1pls1min(PyInt n_orb1, PyInt n_orb2, Double* Ra1, Double* Rcaa1, Double* Rc2, Double* Rcca2, Double* h, Double* V2221, Double* V2111)
	{
	Double H = 0;
	PyInt p2=0;
	for (p2=0;  p2<n_orb2;  p2++)
		{
		PyInt q2=0;
		for ( q2=0;  q2<n_orb2;  q2++)
			{
			PyInt r2=0;
			for ( r2=0;  r2<n_orb2;  r2++)
				{
				PyInt s1=0;
				for ( s1=0;  s1<n_orb1;  s1++)
					{
					H += V2221[((p2*n_orb2 + q2)*n_orb2 + r2)*n_orb1 + s1] * Rcca2[(q2*n_orb2 + p2)*n_orb2 + r2] * Ra1[s1];
					}
				}
			}
		}
	PyInt p22=0;
	for (p22=0;  p22<n_orb2;  p22++)
		{
		PyInt q1=0;
		for ( q1=0;  q1<n_orb1;  q1++)
			{
			PyInt r1=0;
			for (r1=0;  r1<n_orb1;  r1++)
				{
				PyInt s2=0;
				for (s2=0;  s2<n_orb1;  s2++)
					{
					H += V2111[((p22*n_orb1 + q1)*n_orb1 + r1)*n_orb1 + s2] * Rc2[p22] * Rcaa1[(q1*n_orb1 + s2)*n_orb1 + r1];
					}
				}
			}
		}
	H *= 2;
	PyInt p3=0;
	for (p3=0;  p3<n_orb2;  p3++)
		{
		PyInt q3=0;
		for (q3=0;  q3<n_orb1;  q3++)
			{
			H += h[p3*n_orb1 + q3] * Rc2[p3] * Ra1[q3];
			}
		}
	return H;
	}



Double dimer_2pls2min(PyInt n_orb1, PyInt n_orb2, Double* Raa1, Double* Rcc2, Double* V)
	{
	Double H = 0;
	PyInt p2=0;
	for (p2=0;  p2<n_orb2;  p2++)
		{
		PyInt q2=0;
		for (q2=0;  q2<n_orb2;  q2++)
			{
			PyInt r1=0;
			for (r1=0;  r1<n_orb1;  r1++)
				{
				PyInt s1=0;
				for (s1=0;  s1<n_orb1;  s1++)
					{
					H += V[((p2*n_orb2 + q2)*n_orb1 + r1)*n_orb1 + s1] * Rcc2[p2*n_orb2 + q2] * Raa1[s1*n_orb1 + r1];
					}
				}
			}
		}
	return H;
	}


Double dimer_00(PyInt n_orb1, PyInt n_orb2, Double* Rca1, Double* Rca2, Double* V)
	{
	Double H = 0;
	PyInt p1=0;
	for ( p1=0;  p1<n_orb1;  p1++)
		{
		PyInt q2=0;
		for ( q2=0;  q2<n_orb2;  q2++)
			{
			PyInt r1=0;
			for ( r1=0;  r1<n_orb1;  r1++)
				{
				PyInt s2=0;
				for ( s2=0;  s2<n_orb2;  s2++)
					{
					H += V[((p1*n_orb2 + q2)*n_orb1 + r1)*n_orb2 + s2] * Rca1[p1*n_orb1 + r1] * Rca2[q2*n_orb2 + s2];
					}
				}
			}
		}
	return 4*H;
	}





void dimer_1min1pls_loop(PyInt n_elem, Double** H, PyInt* i, PyInt* j, PyInt* dim, PyFloat* sign, PyInt* n_orb1, PyInt* n_orb2, Double** Rc1,  Double** Rcca1, Double** Ra2, Double** Rcaa2, Double** h, Double** V1, Double** V2)
{
    
	// define  iterator variables
    int n;
    int p1,r1,q1,s2;
    int p11,q2,r2;
    int p12,q22;

    // Data divides into four parts 

    int first_q=n_elem/4;
    int sec_q=first_q*2;
    int third_q=first_q*3;
    
    
    /* First quarter data */

	for(n=0; n<first_q; n++) {
        int index = i[n]*dim[n]+j[n];
        double tmp = 0.0;

    
        for(p1=0; p1<n_orb1[n]; p1++) {
            for(r1=0; r1<n_orb1[n]; r1++) {
                for(q1=0; q1<n_orb1[n]; q1++) {
                    for( s2=0; s2<n_orb2[n]; s2++) {
                        tmp += V1[n][((p1*n_orb1[n] + q1)*n_orb1[n] + r1)*n_orb2[n] + s2] * Rcca1[n][(q1*n_orb1[n] + p1)*n_orb1[n] + r1] * Ra2[n][s2];
                    }
                }
            }
        }

    
        for(p11=0; p11<n_orb1[n]; p11++) {
            for(q2=0; q2<n_orb2[n]; q2++) {
                for(r2=0; r2<n_orb2[n]; r2++) {
                    for(s2=0; s2<n_orb2[n]; s2++) {
                        tmp += V2[n][((p11*n_orb2[n] + q2)*n_orb2[n] + r2)*n_orb2[n] + s2] * Rc1[n][p11] * Rcaa2[n][(q2*n_orb2[n] + s2)*n_orb2[n] + r2];
                    }
                }
            }
        }

        tmp *= 2;

        
        for(p12=0; p12<n_orb1[n]; p12++) {
            for(q22=0; q22<n_orb2[n]; q22++) {
                tmp += h[n][p12*n_orb2[n]+q22] * Rc1[n][p12] * Ra2[n][q22];
            }
        }
        H[n][index] = tmp*sign[n];
    }
    
	/*  2nd Quarter*/
	

	for(n=first_q; n<sec_q; n++) {
        int index = i[n]*dim[n]+j[n];
        double tmp = 0.0;

        
        for(p1=0; p1<n_orb1[n]; p1++) {
            for(r1=0; r1<n_orb1[n]; r1++) {
                for(q1=0; q1<n_orb1[n]; q1++) {
                    for( s2=0; s2<n_orb2[n]; s2++) {
                        tmp += V1[n][((p1*n_orb1[n] + q1)*n_orb1[n] + r1)*n_orb2[n] + s2] * Rcca1[n][(q1*n_orb1[n] + p1)*n_orb1[n] + r1] * Ra2[n][s2];
                    }
                }
            }
        }

        
        for(p11=0; p11<n_orb1[n]; p11++) {
            for(q2=0; q2<n_orb2[n]; q2++) {
                for(r2=0; r2<n_orb2[n]; r2++) {
                    for(s2=0; s2<n_orb2[n]; s2++) {
                        tmp += V2[n][((p11*n_orb2[n] + q2)*n_orb2[n] + r2)*n_orb2[n] + s2] * Rc1[n][p11] * Rcaa2[n][(q2*n_orb2[n] + s2)*n_orb2[n] + r2];
                    }
                }
            }
        }

        tmp *= 2;

        
        for(p12=0; p12<n_orb1[n]; p12++) {
            for(q22=0; q22<n_orb2[n]; q22++) {
                tmp += h[n][p12*n_orb2[n]+q22] * Rc1[n][p12] * Ra2[n][q22];
            }
        }
        H[n][index] = tmp*sign[n];
    }



	/* Third Quarter */

	for(n=sec_q; n<third_q; n++) {
        int index = i[n]*dim[n]+j[n];
        double tmp = 0.0;

        
        for(p1=0; p1<n_orb1[n]; p1++) {
            for(r1=0; r1<n_orb1[n]; r1++) {
                for(q1=0; q1<n_orb1[n]; q1++) {
                    for( s2=0; s2<n_orb2[n]; s2++) {
                        tmp += V1[n][((p1*n_orb1[n] + q1)*n_orb1[n] + r1)*n_orb2[n] + s2] * Rcca1[n][(q1*n_orb1[n] + p1)*n_orb1[n] + r1] * Ra2[n][s2];
                    }
                }
            }
        }

        
        for(p11=0; p11<n_orb1[n]; p11++) {
            for(q2=0; q2<n_orb2[n]; q2++) {
                for(r2=0; r2<n_orb2[n]; r2++) {
                    for(s2=0; s2<n_orb2[n]; s2++) {
                        tmp += V2[n][((p11*n_orb2[n] + q2)*n_orb2[n] + r2)*n_orb2[n] + s2] * Rc1[n][p11] * Rcaa2[n][(q2*n_orb2[n] + s2)*n_orb2[n] + r2];
                    }
                }
            }
        }

        tmp *= 2;

       
        for(p12=0; p12<n_orb1[n]; p12++) {
            for(q22=0; q22<n_orb2[n]; q22++) {
                tmp += h[n][p12*n_orb2[n]+q22] * Rc1[n][p12] * Ra2[n][q22];
            }
        }
        H[n][index] = tmp*sign[n];
    }




	/* Fourth Quarter */

        for(n=third_q; n<n_elem; n++) {
        int index = i[n]*dim[n]+j[n];
        double tmp = 0.0;

        
        for(p1=0; p1<n_orb1[n]; p1++) {
            for(r1=0; r1<n_orb1[n]; r1++) {
                for(q1=0; q1<n_orb1[n]; q1++) {
                    for( s2=0; s2<n_orb2[n]; s2++) {
                        tmp += V1[n][((p1*n_orb1[n] + q1)*n_orb1[n] + r1)*n_orb2[n] + s2] * Rcca1[n][(q1*n_orb1[n] + p1)*n_orb1[n] + r1] * Ra2[n][s2];
                    }
                }
            }
        }

        
        for(p11=0; p11<n_orb1[n]; p11++) {
            for(q2=0; q2<n_orb2[n]; q2++) {
                for(r2=0; r2<n_orb2[n]; r2++) {
                    for(s2=0; s2<n_orb2[n]; s2++) {
                        tmp += V2[n][((p11*n_orb2[n] + q2)*n_orb2[n] + r2)*n_orb2[n] + s2] * Rc1[n][p11] * Rcaa2[n][(q2*n_orb2[n] + s2)*n_orb2[n] + r2];
                    }
                }
            }
        }

        tmp *= 2;

        
        for(p12=0; p12<n_orb1[n]; p12++) {
            for(q22=0; q22<n_orb2[n]; q22++) {
                tmp += h[n][p12*n_orb2[n]+q22] * Rc1[n][p12] * Ra2[n][q22];
            }
        }
        H[n][index] = tmp*sign[n];
    }



    
}


void monomer_loop(PyInt n_elem, Double** H, PyInt* i, PyInt* j, PyInt* dim, PyInt* n_orb, Double** Rca,  Double** Rccaa, Double** h, Double** V)
	{
	PyInt n=0;
	for ( n=0;  n<n_elem;  n++)
		{
		PyInt index = i[n]*dim[n] + j[n];
		H[n][index] = monomer(n_orb[n], Rca[n], Rccaa[n], h[n], V[n]);
		}
	return;
	}

void dimer_2min2pls_loop(PyInt n_elem, Double** H, PyInt* i, PyInt* j, PyInt* dim, PyInt* n_orb1, PyInt* n_orb2, Double** Rcc1, Double** Raa2, Double** V)
	{
	PyInt n=0;
	for ( n=0;  n<n_elem;  n++)
		{
		PyInt index = i[n]*dim[n] + j[n];
		H[n][index] = dimer_2min2pls(n_orb1[n], n_orb2[n], Rcc1[n], Raa2[n], V[n]);
		}
	return;
	}


void dimer_00_loop(PyInt n_elem, Double** H, PyInt* i, PyInt* j, PyInt* dim, PyInt* n_orb1, PyInt* n_orb2, Double** Rca1, Double** Rca2, Double** V)
	{
	PyInt n=0;
	for ( n=0;  n<n_elem;  n++)
		{
		PyInt index = i[n]*dim[n] + j[n];
		H[n][index] = dimer_00(n_orb1[n], n_orb2[n], Rca1[n], Rca2[n], V[n]);
		}
	return;
	}

void dimer_1pls1min_loop(PyInt n_elem, Double** H, PyInt* i, PyInt* j, PyInt* dim, PyFloat* sign, PyInt* n_orb1, PyInt* n_orb2, Double** Ra1,  Double** Rcaa1, Double** Rc2, Double** Rcca2, Double** h, Double** V1, Double** V2)
	{
	PyInt n=0;
	for ( n=0;  n<n_elem;  n++)
		{
		PyInt index = i[n]*dim[n] + j[n];
		H[n][index] = sign[n] * dimer_1pls1min(n_orb1[n], n_orb2[n], Ra1[n], Rcaa1[n], Rc2[n], Rcca2[n], h[n], V1[n], V2[n]);
		}
	return;
	}

void dimer_2pls2min_loop(PyInt n_elem, Double** H, PyInt* i, PyInt* j, PyInt* dim, PyInt* n_orb1, PyInt* n_orb2, Double** Raa1, Double** Rcc2, Double** V)
	{
	PyInt n=0;
	for ( n=0;  n<n_elem;  n++)
		{
		PyInt index = i[n]*dim[n] + j[n];
		H[n][index] = dimer_2pls2min(n_orb1[n], n_orb2[n], Raa1[n], Rcc2[n], V[n]);
		}
	return;
	}


}
