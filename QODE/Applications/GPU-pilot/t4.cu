#include <hip/hip_runtime.h>

#include "stdio.h"
#include <sys/time.h>
#define DEBUG 1

extern "C" {

  // cleaner error handling; just wrap cuda library calls with gpuErrchk(foo());
  #define gpuErr(ans) { gpuAssert((ans), __FILE__, __LINE__); }
  inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
  {
    if (code != hipSuccess) 
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
  }

  __device__ int get_tid()
  {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x
      + blockIdx.z * gridDim.x * gridDim.y;
    return  blockId * (blockDim.x * blockDim.y * blockDim.z)
      + (threadIdx.z * (blockDim.x * blockDim.y))
      + (threadIdx.y * blockDim.x)
      + threadIdx.x;
  }

  __global__ void dimer_1min1pls_t4(int n_elem, double *d_H, int *d_i, int *d_j, int *d_dim, float *d_sign, int *d_n_orb1, int *d_n_orb2, double *d_Rc1, double *d_Rcca1, double *d_Ra2, double *d_Rcaa2, double *d_h, double *d_V1, double *d_V2)
  {
    const int tid = get_tid();
    if(tid < n_elem) {
      const int n_orb1 = d_n_orb1[tid];
      const int n_orb2 = d_n_orb2[tid];
      double tmp = 0.0;
      int p,q,r,s;

      // In practice the n calculated here is not equal to the "actual" n here
      // since multiple kernel calls will be needed.
      // n_elem is the length

      // Top Loop
      // Use the tid to calculate variables in five dimensions
      //int n = tid;
      s = tid % n_orb2;
      r = ((tid - s) / (n_orb2)) % n_orb1;
      q = ((tid - r - s) * (n_orb2 * n_orb1)) % n_orb1;
      p = ((tid - q - r - s) / (n_orb2 * n_orb1 * n_orb1)) % n_orb1;
      tmp = d_V1[(((p * n_orb1 + q ) * n_orb1 + r) * n_orb2 + s) * n_elem + tid]
        * d_Rcca1[((q * n_orb1 + p)* n_orb1 + r) * n_elem + tid]
        * d_Ra2[s * n_elem + tid];

      // Middle Loop
      // 5d calculation
      s = tid % n_orb2;
      r = ((tid - s) / (n_orb2)) % n_orb2;
      q = ((tid - r - s) * (n_orb2 * n_orb2)) % n_orb2;
      p = ((tid - q - r - s) / (n_orb2 * n_orb2 * n_orb2)) % n_orb1;

      tmp += d_V2[(((p * n_orb2 + q ) * n_orb2 + r) * n_orb2 + s) * n_elem + tid]
        * d_Rcaa2[((q * n_orb2 + s) * n_orb2 + r) * n_elem + tid]
        * d_Rc1[p * n_elem + tid];
      tmp *= 2.0;

      // Bottom loop
      // 3d calculation
      s = tid % n_orb2;
      r = ((tid - s) / (n_orb2)) % n_orb1;
      tmp += d_h[(r * n_orb2 * s) * n_elem + tid]
        * d_Rc1[r * n_elem + tid]
        * d_Ra2[s * n_elem + tid];
      tmp *= d_sign[tid];

      // Assignment
      int index = d_i[tid] * d_dim[tid] + d_j[tid];
      d_H[index * n_elem + tid] = tmp;

      }
  }

void dimer_1min1pls_loop(int n_elem, double** H, int* i, int* j, int* dim, float* sign, int* n_orb1, int* n_orb2, double** Rc1,  double** Rcca1, double** Ra2, double** Rcaa2, double** h, double** V1, double** V2)
{
    struct timeval start,stop; 
    gettimeofday(&start,0);

    double *d_H, *d_Rc1, *d_Rcca1, *d_Ra2, *d_Rcaa2, *d_h, *d_V1, *d_V2;
    int *d_i, *d_j, *d_n_orb1, *d_n_orb2, *d_dim;
    float *d_sign;

    const int n1 = n_orb1[0];
    const int n2 = n_orb2[0];

    const int V1_len = n_elem * n2 * n1 * n1 * n1;
    const int V2_len = n_elem * n2 * n2 * n2 * n1;
    const int Rc1_len = n_elem * n1;
    const int Ra2_len = n_elem * n2;
    const int Rcca1_len = n_elem * n1 * n1 *n1;
    const int Rcaa2_len = n_elem * n2 * n2 * n2;
    const int h_len = n_elem * n1 * n2;
    const int H_len = n_elem * n1 * n1 * n1 * n2; // could be an issue if n1 != n2


    const int chunks = n1 * n2; // for maximum thread utilization this should go evenly into H_len
    const int elemPerChunk = ceil(n_elem / chunks);
    const int threadsPerChunk = ceil(H_len / chunks);
    const dim3 dimblock(4, 4, 4);
    const dim3 dimgrid(ceil(threadsPerChunk / dimblock.x), ceil(threadsPerChunk / dimblock.y), ceil(threadsPerChunk / dimblock.z));

    if(DEBUG) {
      printf("Chunks: %d\tThreads per chunk: %d\n",chunks,threadsPerChunk);
      printf("dimblock:\t<%d\t%d\t%d>\ndimgrid:\t<%d\t%d\t%d>\n",dimblock.x,dimblock.y,dimblock.z,dimgrid.x,dimgrid.y,dimgrid.z);
    }

    const int V1_size = sizeof(double) * ceil(V1_len / chunks);
    const int V2_size = sizeof(double) * ceil(V2_len / chunks);
    const int Rc1_size = sizeof(double) * ceil(Rc1_len / chunks);
    const int Ra2_size = sizeof(double) * ceil(Ra2_len / chunks);
    const int Rcca1_size = sizeof(double) * ceil(Rcca1_len / chunks);
    const int Rcaa2_size = sizeof(double) * ceil(Rcaa2_len / chunks);
    const int h_size = sizeof(double) * ceil(h_len / chunks);
    const int i_size = sizeof(int) * elemPerChunk; // for all the n-elem int arrays
    const int sign_size = sizeof(float) * elemPerChunk;
    const int H_size = sizeof(double) * threadsPerChunk;

    if(DEBUG) {
      printf("Starting device memory allocation\n");
    }

    gpuErr(hipMalloc((void **) &d_H,       H_size));
    gpuErr(hipMalloc((void **) &d_Rc1,     Rc1_size));
    gpuErr(hipMalloc((void **) &d_V1,      V1_size));
    gpuErr(hipMalloc((void **) &d_V2,      V2_size));
    gpuErr(hipMalloc((void **) &d_Ra2,     Ra2_size));
    gpuErr(hipMalloc((void **) &d_Rcca1,   Rcca1_size));
    gpuErr(hipMalloc((void **) &d_Rcaa2,   Rcaa2_size));
    gpuErr(hipMalloc((void **) &d_h,       h_size));
    gpuErr(hipMalloc((void **) &d_i,       i_size));
    gpuErr(hipMalloc((void **) &d_j,       i_size));
    gpuErr(hipMalloc((void **) &d_dim,     i_size));
    gpuErr(hipMalloc((void **) &d_n_orb1,  i_size));
    gpuErr(hipMalloc((void **) &d_n_orb2,  i_size));
    gpuErr(hipMalloc((void **) &d_sign,    sign_size));

    if(DEBUG) {
      printf("Finished device memory allocation\n");
    }

    for(int k=0; k<chunks; k++) {

      if(DEBUG) {
        printf("Chunk %d: starting host to device memcpys\n",k);
      }

      gpuErr(hipMemcpy(d_Rc1, Rc1 + k * Rc1_len, Rc1_size, hipMemcpyHostToDevice));
      gpuErr(hipMemcpy(d_V1, V1 + k * V1_len, V1_size, hipMemcpyHostToDevice));
      gpuErr(hipMemcpy(d_V2, V2 + k * V2_len , V2_size, hipMemcpyHostToDevice));
      gpuErr(hipMemcpy(d_Ra2, Ra2 + k * Ra2_len, Ra2_size, hipMemcpyHostToDevice));
      gpuErr(hipMemcpy(d_Rcca1, Rcca1 + k * Rcca1_len, Rcca1_size, hipMemcpyHostToDevice));
      gpuErr(hipMemcpy(d_Rcaa2, Rcaa2 + k * Rcaa2_len, Rcaa2_size, hipMemcpyHostToDevice));
      gpuErr(hipMemcpy(d_h, h + k * h_len, h_size, hipMemcpyHostToDevice));
      gpuErr(hipMemcpy(d_i, i + k * elemPerChunk, i_size, hipMemcpyHostToDevice));
      gpuErr(hipMemcpy(d_j, j + k * elemPerChunk, i_size, hipMemcpyHostToDevice));
      gpuErr(hipMemcpy(d_n_orb1, n_orb1 + k * elemPerChunk, i_size, hipMemcpyHostToDevice));
      gpuErr(hipMemcpy(d_n_orb2, n_orb2 + k * elemPerChunk, i_size, hipMemcpyHostToDevice));
      gpuErr(hipMemcpy(d_sign, sign + k * elemPerChunk, sign_size, hipMemcpyHostToDevice));

      if(DEBUG) {
        printf("Chunk %d: finished host to device memcpys\n",k);
        printf("Chunk %d: launching dimer_1min1pls_t4 kernel\n",k);
      }

      dimer_1min1pls_t4<<<dimblock,dimgrid>>>(n_elem,d_H,d_i,d_j,d_dim,d_sign,d_n_orb1,d_n_orb2,d_Rc1,d_Rcca1,d_Ra2,d_Rcaa2,d_h,d_V1,d_V2);
      gpuErr(hipPeekAtLastError());
      gpuErr(hipDeviceSynchronize());

      memcpy(H+k*H_len, d_H, H_size);


      if(DEBUG) {
        printf("Chunk %d: finished executing dimer_1min1pls_t4 kernel \n",k);
      }


    }

    gpuErr(hipFree(d_Rc1));
    gpuErr(hipFree(d_V1));
    gpuErr(hipFree(d_V2));
    gpuErr(hipFree(d_Ra2));
    gpuErr(hipFree(d_Rcca1));
    gpuErr(hipFree(d_Rcaa2));
    gpuErr(hipFree(d_h));
    gpuErr(hipFree(d_i));
    gpuErr(hipFree(d_j));
    gpuErr(hipFree(d_n_orb1));
    gpuErr(hipFree(d_n_orb2));
    gpuErr(hipFree(d_sign))
    gettimeofday(&stop,0);
    if(DEBUG) {
      double t = (double)(stop.tv_sec-start.tv_sec)*1000+(double)(stop.tv_usec-start.tv_usec)/1000;
      printf("dimer_1min1pls_loop_t4 finished in %f ms\n", t);
    }


}
}
