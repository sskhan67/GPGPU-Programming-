#include<stdio.h>
#include<string.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<time.h>
#include<sys/time.h>
#include<math.h>
int main(int argc, char ** argv){


	float *host, *device; 
	struct timeval start_timeS,end_timeE;
	int size;
	
	//int a = atof(argv[1]);	
	int a[]={9400,26200,59000,1,2,4,8,16,32,64,128,256,512,1000,2000,4000,8000,16000,32000,64000,128000,256000,512000};
	

//        int a[]={1,2,4,8,16,32,64,128,256,512,1000,2000,4000,8000,16000,32000,64000,128000,256000,512000};

	FILE *fp ;
        fp=fopen("d2h.csv","w+");
        fprintf(fp,"KB, time, BW");
	
for (int k=0;k<20;k++)
{	
	size = a[k]*1024; 
	host = (float*)malloc(sizeof(float)*size);
	hipMalloc((void**)&device,sizeof(float)*size);

	for (int i = 0; i<size/4;i++){
		host[i]=rand();
	}


	hipMemcpy(device,host,sizeof(float)*size,hipMemcpyHostToDevice);

	hipDeviceSynchronize();



    gettimeofday(&start_timeS,NULL);
    for (int i = 0; i<30; i++){
    hipMemcpy(host,device,sizeof(float)*size,hipMemcpyDeviceToHost);
    }
    gettimeofday(&end_timeE,NULL);

    hipDeviceSynchronize();


	float timeM = ((end_timeE.tv_sec*1000000+end_timeE.tv_usec)-(start_timeS.tv_sec*1000000+start_timeS.tv_usec));
	float t=timeM/(1000*30);// time in ms
	float BW=a[k]/(t/1000); // kbps	
	printf("KB: %d, time from device to host : %f ms, BW[kpbs]: %f \n",a[k],t,BW);
	fprintf(fp,"\n%d, %f, %f \n",a[k],t,BW);

}
}
