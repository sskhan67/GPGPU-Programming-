#include "hip/hip_runtime.h"

//author : Syeduzzaman Ikhan
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include "image_template.h"
#include "convolution.h"
#include"gaussian_kernel.h"
#include"gradient.h"
#include"suppress.h"
#include"hysteresis.h"
#include"edge.h"
#include"cornerness.h"
#include "sys/time.h"
#include <hip/hip_runtime.h>
#include<thrust/host_vector.h>
#include<thrust/device_vector.h>
#include<thrust/sort.h>


/*
Project Goal: To implement Canny Edge detection's 7 Feature detection and Parallelization using Open MP 

I/O File :  Test Image
Main function: Takes two argument as image path and sigma value=0.6 or 1 or 1.25

Setps: 

	User defined Header Files: image.h -> Image I/O handler, gaussian_kernel.h-> create gaussian kernel using user defined sigma value =0.6 or 1 or 1.25, convolution.h-> To perform
	convolution using Gaussian and Gaussian derivative kernel,  gradient.h-> perform horizontal and vertical gradient , boundary.h-> to handle boudary for convolution  
 hystersis+ suppress+edges 



*/


// main function starts here 
int main(int argc, char **argv)
{
		struct timeval start,end,start_k1,end_k1,start_k2,end_k2,start_k3,end_k3,start_k4,end_k4,start_k5,end_k5,start_c1,end_c1,start_c2,end_c2,start_c3,end_c3,start_nf,end_nf,start_f1,end_f1,start_f2,end_f2;

		
		// Host variable 
		float *image,*g_kernel, *deriv_kernel, *temp_hor, *temp_ver,  *horizontal_grad, *vertical_grad, *magnitude, *phase,*sup,*hyst,*edge,*corner,*feature;	//  create pointer variable 
		int height, width, k_width;// image height, width, and hernel width size 
		float sigma=atof(argv[2]);
		int a=round(2.5*sigma-0.5);
		k_width=2*a+1;
		
			//global memory access variable

		float *flop,*global,*flop1,*global1,*global2,*flop2,*flop3,*global4,*global_h;

		// Device variable 
float *d_image,*d_temp_hor,*d_horizontal_grad,*d_temp_ver,*d_vertical_grad,*d_magnitude,*d_phase,*d_sup,*d_hyst,*d_edge,*d_corner,*d_feature;
float *d_kernel,*d_deriv_kernel;

float *d_flop,*d_global,*d_flop1,*d_global1,*d_global2,*d_flop2,*d_flop3,*d_global3,*d_global_h;

// with file I/O time start
gettimeofday(&start,NULL);



// start file I/O time 
gettimeofday(&start_f1,NULL);

        	// step 1. call function to read image and pass the funtion to the function 
		read_image_template(argv[1],&image,&width,&height);
// end time for File I/O
gettimeofday(&end_f1,NULL);



//Malloc for Host 
temp_hor = (float*)malloc(sizeof(float)*height*width);
horizontal_grad = (float*)malloc(sizeof(float)*height*width);
temp_ver = (float*)malloc(sizeof(float)*height*width);
vertical_grad = (float*)malloc(sizeof(float)*height*width);
magnitude = (float*)malloc(sizeof(float)*height*width);
phase = (float*)malloc(sizeof(float)*height*width);
sup = (float*)malloc(sizeof(float)*height*width);
hyst = (float*)malloc(sizeof(float)*height*width);
edge = (float*)malloc(sizeof(float)*height*width);
corner = (float*)malloc(sizeof(float)*height*width);
feature = (float*)malloc(sizeof(float)*height*width);

flop = (float*)malloc(sizeof(float)*height*width);
global = (float*)malloc(sizeof(float)*height*width);

flop1 = (float*)malloc(sizeof(float)*height*width);
global1 = (float*)malloc(sizeof(float)*height*width);

flop2= (float*)malloc(sizeof(float)*height*width);
global2 = (float*)malloc(sizeof(float)*height*width);

flop3= (float*)malloc(sizeof(float)*height*width);
global4 = (float*)malloc(sizeof(float)*height*width);
global_h = (float*)malloc(sizeof(float)*height*width);

		
		// Malloc for DEVICE GPU
hipMalloc((void **)&d_image,sizeof(float)*width*height);
hipMalloc((void **)&d_temp_hor,sizeof(float)*width*height);
hipMalloc((void **)&d_horizontal_grad,sizeof(float)*width*height);
hipMalloc((void **)&d_temp_ver,sizeof(float)*width*height);
hipMalloc((void **)&d_vertical_grad,sizeof(float)*width*height);
hipMalloc((void **)&d_magnitude,sizeof(float)*width*height);
hipMalloc((void **)&d_phase,sizeof(float)*width*height);
hipMalloc((void **)&d_sup,sizeof(float)*width*height);
hipMalloc((void **)&d_hyst,sizeof(float)*width*height);
hipMalloc((void **)&d_edge,sizeof(float)*width*height);
hipMalloc((void **)&d_corner,sizeof(float)*width*height);
hipMalloc((void **)&d_feature,sizeof(float)*width*height);
//udaMalloc((void **)&d_feature,sizeof(float)*width*height);



hipMalloc((void **)&d_flop,sizeof(float)*width*height);
hipMalloc((void **)&d_global,sizeof(float)*width*height);
hipMalloc((void **)&d_flop1,sizeof(float)*width*height);
hipMalloc((void **)&d_global1,sizeof(float)*width*height);
hipMalloc((void **)&d_flop2,sizeof(float)*width*height);
hipMalloc((void **)&d_global2,sizeof(float)*width*height);
hipMalloc((void **)&d_flop3,sizeof(float)*width*height);
hipMalloc((void **)&d_global3,sizeof(float)*width*height);
hipMalloc((void **)&d_global_h,sizeof(float)*width*height);




hipMalloc((void **)&d_kernel,sizeof(float)*k_width);
hipMalloc((void **)&d_deriv_kernel,sizeof(float)*k_width);

		// step 2. Create gaussian kernel and gaussian derivative kernel using sigma 
		
		gaussian(&g_kernel, &deriv_kernel, atof(argv[2]), &k_width); // atof function converts sigma string value to type double 



	// time ->  communication time 

gettimeofday(&start_c1,NULL);
	
	//copy the items from CPU to GPU
hipMemcpy(d_image,image,sizeof(float)*width*height,hipMemcpyHostToDevice);
hipMemcpy(d_kernel,g_kernel,sizeof(float)*k_width,hipMemcpyHostToDevice);
hipMemcpy(d_deriv_kernel,deriv_kernel,sizeof(float)*k_width,hipMemcpyHostToDevice);
hipDeviceSynchronize();
	// end communication time 

gettimeofday(&end_c1,NULL);

float c1=((end_c1.tv_sec*1000000+end_c1.tv_usec)-(start_c1.tv_sec*1000000+start_c1.tv_usec));


//Horizontal
int block_dim=atof(argv[3]);
//int block_dim = 32;
dim3 dmBlock(block_dim, block_dim, 1);
dim3 dmGrid(ceil(height/block_dim), ceil(width/block_dim), 1);


	// kernel time start 
gettimeofday(&start_k1,NULL);

/*
convoultion<<<dmGrid,dmBlock>>>(d_image, d_temp_hor,d_kernel,height,width,k_width,1);
convoultion<<<dmGrid,dmBlock>>>(d_temp_hor,d_horizontal_grad,d_deriv_kernel,height,width,1,k_width);
hipDeviceSynchronize();

*/
convoultion<<<dmGrid,dmBlock,sizeof(float)*block_dim*block_dim>>>(d_image, d_temp_hor,d_kernel,d_flop1,d_global1,height,width,k_width,1);	


	// flop & global memory access calculation
	hipMemcpy(flop1,d_flop1,sizeof(float)*width*height,hipMemcpyDeviceToHost);
	hipDeviceSynchronize();


	hipMemcpy(global1,d_global1,sizeof(float)*width*height,hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	


convoultion<<<dmGrid,dmBlock,sizeof(float)*block_dim*block_dim>>>(d_temp_hor,d_horizontal_grad,d_deriv_kernel,d_flop1,d_global1,height,width,1,k_width); 
hipDeviceSynchronize();



	    
// Vertical
/*
convoultion<<<dmGrid,dmBlock>>>(d_image, d_temp_ver,d_kernel,height,width,k_width,1);
convoultion<<<dmGrid,dmBlock>>>(d_temp_ver,d_vertical_grad,d_deriv_kernel,height,width,1,k_width);

*/

convoultion<<<dmGrid,dmBlock,sizeof(float)*block_dim*block_dim>>>(d_image, d_temp_ver,d_kernel,d_flop1,d_global1,height,width,k_width,1);
convoultion<<<dmGrid,dmBlock,sizeof(float)*block_dim*block_dim>>>(d_temp_ver,d_vertical_grad,d_deriv_kernel,d_flop1,d_global1,height,width,1,k_width);


hipDeviceSynchronize();
	
	//kernel time end

gettimeofday(&end_k1,NULL);

float k1=((end_k1.tv_sec*1000000+end_k1.tv_usec)-(start_k1.tv_sec*1000000+start_k1.tv_usec));
printf("\nHorizontal and Vertical Kernel time(ms): %f\n",k1/1000);

// Magnitude & Phase
 
	// kernel time start
gettimeofday(&start_k2,NULL);

gradient<<<dmGrid,dmBlock>>>(d_vertical_grad,d_horizontal_grad,d_magnitude,d_phase,d_flop2,d_global2,height,width);
hipDeviceSynchronize();


	// flop & global memory access calculation
        hipMemcpy(flop2,d_flop2,sizeof(float)*width*height,hipMemcpyDeviceToHost);
        hipDeviceSynchronize();


        hipMemcpy(global2,d_global2,sizeof(float)*width*height,hipMemcpyDeviceToHost);
        hipDeviceSynchronize();


	// kernel time end
gettimeofday(&end_k2,NULL);

float k2=((end_k2.tv_sec*1000000+end_k2.tv_usec)-(start_k2.tv_sec*1000000+start_k2.tv_usec));
printf("Mag & Phase  Kernel time(ms): %f\n",k2/1000);


// Suppression

	// kernel time start

gettimeofday(&start_k3,NULL);

suppress<<<dmGrid,dmBlock>>>(d_magnitude,d_phase,d_sup,d_flop3,d_global3,height,width);
hipDeviceSynchronize();


	
	// flop & global memory access calculation
        
	
	hipMemcpy(flop3,d_flop3,sizeof(float)*width*height,hipMemcpyDeviceToHost);
        hipDeviceSynchronize();


        hipMemcpy(global4,d_global3,sizeof(float)*width*height,hipMemcpyDeviceToHost);
        hipDeviceSynchronize();




	//kernel time end
gettimeofday(&end_k3,NULL);

float k3=((end_k3.tv_sec*1000000+end_k3.tv_usec)-(start_k3.tv_sec*1000000+start_k3.tv_usec));
printf("Supression Kernel time(ms): %f\n",k3/1000);



thrust::device_ptr<float> thr_d(d_sup);

thrust::device_vector<float>sup_vec(thr_d,thr_d+(height*width));

thrust::sort(sup_vec.begin(),sup_vec.end());

int index = (int)((0.95)*height*width);

float t_high = sup_vec[index];

float t_low =t_high/5;


// hysteresis and edge linking 


	//kernel time start
gettimeofday(&start_k4,NULL);

finaledge<<<dmGrid,dmBlock>>>(d_sup,d_edge,d_hyst,t_high,t_low,d_global_h,height,width);
hipDeviceSynchronize();

	//kernel time end
gettimeofday(&end_k4,NULL);


	 // flop & global memory access calculation


        //hipMemcpy(flop3,d_flop3,sizeof(float)*width*height,hipMemcpyDeviceToHost);
        //hipDeviceSynchronize();


        hipMemcpy(global_h,d_global_h,sizeof(float)*width*height,hipMemcpyDeviceToHost);
        hipDeviceSynchronize();





float k4=((end_k4.tv_sec*1000000+end_k4.tv_usec)-(start_k4.tv_sec*1000000+start_k4.tv_usec));
printf("Hyster. & Final edge Kernel time(ms): %f\n",k4/1000);




	//start communication time
gettimeofday(&start_c2,NULL);


hipMemcpy(edge, d_edge, sizeof(float)*width*height, hipMemcpyDeviceToHost);
hipDeviceSynchronize();

	//end communication time 
gettimeofday(&end_c2,NULL);

float  c2=((end_c2.tv_sec*1000000+end_c2.tv_usec)-(start_c2.tv_sec*1000000+start_c2.tv_usec));


// CORNERESS funtion 

	//kernel time start
gettimeofday(&start_k5,NULL);

feature_detector<<<dmGrid,dmBlock,2*sizeof(float)*block_dim*block_dim>>>(d_corner,d_global,d_flop,height, width, d_vertical_grad, d_horizontal_grad, block_dim);
//hipMemcpy(corner,d_corner,sizeof(float)*width*height,hipMemcpyDeviceToHost);
        
find_featureGPU<<<dmGrid,dmBlock,2*sizeof(float)*block_dim*block_dim>>>(d_feature,d_corner,height, width,block_dim);


 	//kernel time end
gettimeofday(&end_k5,NULL);

float k5=((end_k5.tv_sec*1000000+end_k5.tv_usec)-(start_k5.tv_sec*1000000+start_k5.tv_usec));
printf(" Corner & Feature  detection  Kernel time(ms): %f\n",k5/1000);



	 //start communication time
gettimeofday(&start_c3,NULL);
hipMemcpy(feature,d_feature,sizeof(float)*width*height,hipMemcpyDeviceToHost);
hipDeviceSynchronize();

//end communication time 
gettimeofday(&end_c3,NULL);

float  c3=((end_c3.tv_sec*1000000+end_c3.tv_usec)-(start_c3.tv_sec*1000000+start_c3.tv_usec));


// CSV File writting for Features 
	FILE *fp ;
    	fp=fopen("corner.csv","w+");
	 fprintf(fp,"i,j");
	int LocI,LocJ;
	for (int i=0;i<height*width;i++)
	{
	if (*(feature+i)>0)
		{

		int a=*(feature+i);
		LocI=a/width;
		LocJ=a % width;
		
		fprintf(fp,"\n%d, %d\n",LocI,LocJ);
		}
	}
	// Flop and global for Feature detetector  

hipMemcpy(flop,d_flop,sizeof(float)*width*height,hipMemcpyDeviceToHost);
hipDeviceSynchronize();


hipMemcpy(global,d_global,sizeof(float)*width*height,hipMemcpyDeviceToHost);
hipDeviceSynchronize();


// flops

float sum_ff=0,sum_f2=0,sum_f3,sum_f4;

for(int i=0;i<height*width;i++)
{

	if(*(flop+i)>0)
	{	
	sum_ff=sum_ff+flop[i];
	sum_f2=sum_f2+flop1[i];
	sum_f3=sum_f3+flop2[i];


	}	
}

printf("------Flop Calculation-------\n");
printf("\n Horizontal Flops: %f\n",4*sum_f2);

printf("\n Mag & Phase Flops: %f\n",sum_f3);


printf("\n Feature Flopsa: %f\n",sum_ff);


printf("------Memory access  Calculation-------\n");


// global memory

float sum_gf=0,sum_g2=0,sum_g3=0,sum_g4=0,sum_g5=0;

for(int i=0;i<height*width;i++)
{

        if(*(global+i)>0)
        {
        sum_gf=sum_gf+global[i];
        
        sum_g2=sum_g2+global1[i];
        sum_g3=sum_g3+global2[i];
        //sum_g4=sum_g4+global4[i];
        //sum_g5=sum_g5+global_h[i];



        }
}

printf("Horizontal global : %f\n",4*sum_g2);
printf("Mag & Phase Global : %f\n",sum_g3);
printf("feature_global: %f\n\n",sum_gf);


printf("-------------\n");







	// File I/O time start
gettimeofday(&start_f2,NULL);



write_image_template("edge.pgm",edge, width, height);

	// file i/o time over 
gettimeofday(&end_f2,NULL);


	// end to end time
gettimeofday(&end,NULL);

float time_io=((end_f1.tv_sec*1000000+end_f1.tv_usec)-(start_f1.tv_sec*1000000+start_f1.tv_usec))+   ((end_f2.tv_sec*1000000+end_f2.tv_usec)-(start_f2.tv_sec*1000000+start_f2.tv_usec));
float time_k=(k1+k2+k3+k4)/1000;

float time_c=(c1+c2+c3)/1000;
float time_t1=((end.tv_sec*1000000+end.tv_usec)-(start.tv_sec*1000000+start.tv_usec));
printf("Image width: %d,Image Height: %d, Sigma: %f,Block size: %d, Kernel time (ms): %f, communication time (ms): %f, Parallel time (ms): %f, file i/o time: %f, end-to-end time with file i/o: %f\n",width,height,sigma,block_dim,time_k,time_c,time_k+time_c,time_io/1000,time_t1/1000);

printf("C1: %f",c1/1000);
printf("C2: %f",c2/1000);

	hipFree(image);
 	hipFree(d_image);
	hipFree(d_temp_hor);
	hipFree(d_horizontal_grad);
	hipFree(d_temp_ver);
    	hipFree(d_vertical_grad);
	hipFree(d_magnitude);
	hipFree(d_phase);	
	hipFree(sup);
	hipFree(hyst);
	hipFree(edge);
	hipFree(d_corner);
	hipFree(d_feature);
        hipFree(d_global);
        hipFree(d_global1);
        hipFree(d_global2);
        //hipFree(d_global4);

        hipFree(d_flop);
        hipFree(d_flop1);
        hipFree(d_flop2);



// Tp= communication time +kernel time 
// Ts= compution time without File I/O 

}



