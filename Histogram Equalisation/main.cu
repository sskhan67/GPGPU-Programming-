#include "hip/hip_runtime.h"

//author : Syeduzzaman Ikhan
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include "image_template.h"
#include "histogramEqualisation.h"
#include<hip/hip_runtime.h>
#include "sys/time.h"

// main function starts here 
int main(int argc, char **argv)
{
		
		

		// Host variable 
		float *image,*temp,*h_hist,*h_cdf;	//  create pointer variable 
		int height, width;// image height, width, and hernel width size 
		
		// device variable
		float *d_image,*d_temp,*d_hist,*d_cdf;
		
// step 1. call function to read image and pass the funtion to the function 
                read_image_template(argv[1],&image,&width,&height);



		//Malloc for Host 
		temp = (float*)malloc(sizeof(float)*height*width);
		h_hist=(float*)malloc(sizeof(float)*256);
		h_cdf=(float*)malloc(sizeof(float)*256);
		
		

		// Malloc for DEVICE GPU
		hipMalloc((void **)&d_image,sizeof(float)*width*height);
		hipMalloc((void **)&d_temp,sizeof(float)*width*height);
		hipMalloc((void **)&d_hist,sizeof(float)*256);
		hipMalloc((void **)&d_cdf,sizeof(float)*256);

		hipMemset(d_hist, 0, 256*sizeof(float));		
		hipMemset(d_cdf, 0, 256*sizeof(float));



        	//copy the items from CPU to GPU
		hipMemcpy(d_image,image,sizeof(float)*width*height,hipMemcpyHostToDevice);
		int block_dim = 16;
		dim3 dmBlock(block_dim, block_dim, 1);
		dim3 dmGrid(ceil(height/block_dim), ceil(width/block_dim), 1);
		





		// global memory Kernel , histrogram 
		hist1<<<dmGrid,dmBlock>>>(d_image,d_hist,width,height);
		hipDeviceSynchronize();
		
		
	//normalization 
		
	for (int i=0;i<256;i++)
		{	
			
        		
			h_hist[i]=h_hist[i]/(height*width);
	
			
		}
		
		h_cdf[0]=h_hist[0];

		//CDF calculation 

		for (int i=1;i<256;i++)
		{
        		h_cdf[i]=h_cdf[i-1]+h_hist[i];
	
		}
	
 		hipMemcpy(d_cdf,h_cdf,sizeof(float)*256,hipMemcpyHostToDevice);
		
		


		//Hist Normalization kernel 
		hist_f<<<dmGrid,dmBlock>>>(d_image,d_cdf,width,height);
        hipDeviceSynchronize();






		hipMemcpy(temp,d_image,sizeof(float)*width*height,hipMemcpyDeviceToHost);


		// write image on disk 

		write_image_template("histrogram.pgm",temp, width, height);

		hipFree(image);
		hipFree(d_image);
		hipFree(temp);
		hipFree(d_temp);


}



